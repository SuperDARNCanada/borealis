
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <vector>

#define NUMELEMENTS 64
__device__ inline hipComplex __shfl_down(hipComplex var, unsigned int srcLane, int width=32){
    float2 a = *reinterpret_cast<float2*>(&var);
    a.x = __shfl_down(a.x, srcLane, width);
    a.y = __shfl_down(a.y, srcLane, width);
    return *reinterpret_cast<hipComplex*>(&a);
}

__device__ hipComplex parallel_reduce(hipComplex* data, int tap_offset) {

    auto filter_tap_num = threadIdx.x;
    auto num_filter_taps = blockDim.x;
    hipComplex total_sum = data[tap_offset];


    if ((num_filter_taps >= 512) && (filter_tap_num < 256))
    {
        data[tap_offset] = total_sum = hipCaddf(total_sum,data[tap_offset + 256]);
    }

    __syncthreads();

    if ((num_filter_taps >= 256) && (filter_tap_num < 128))
    {
            data[tap_offset] = total_sum = hipCaddf(total_sum, data[tap_offset + 128]);
    }

     __syncthreads();

    if ((num_filter_taps >= 128) && (filter_tap_num <  64))
    {
       data[tap_offset] = total_sum = hipCaddf(total_sum, data[tap_offset +  64]);
    }

    __syncthreads();

    if ( filter_tap_num < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (num_filter_taps >=  64) total_sum = hipCaddf(total_sum, data[tap_offset + 32]);
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            total_sum = hipCaddf(total_sum,__shfl_down(total_sum, offset));
        }
    }

    return total_sum;
}

__device__ hipComplex parallel_reduce2(hipComplex* data, int tap_offset){


    for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
        if (threadIdx.x < s)
            data[tap_offset] = hipCaddf(data[tap_offset],
                                                    data[tap_offset + s]);
        __syncthreads();
    }
    if (threadIdx.x < 32){
        data[tap_offset] = hipCaddf(data[tap_offset],
                                                data[tap_offset + 32]);
        __syncthreads();
        data[tap_offset] = hipCaddf(data[tap_offset],
                                                data[tap_offset + 16]);
        __syncthreads();
        data[tap_offset] = hipCaddf(data[tap_offset],
                                                data[tap_offset + 8]);
        __syncthreads();
        data[tap_offset] = hipCaddf(data[tap_offset],
                                                data[tap_offset + 4]);
        __syncthreads();
        data[tap_offset] = hipCaddf(data[tap_offset],
                                                data[tap_offset + 2]);
        __syncthreads();
        data[tap_offset] = hipCaddf(data[tap_offset],
                                                data[tap_offset + 1]);
        __syncthreads();
    }

    return data[0];
}

__global__ void add_numbers(hipComplex* data, hipComplex* reduced_sum){
    extern __shared__ hipComplex shr_data[];

    shr_data[threadIdx.x] = data[threadIdx.x];
    __syncthreads();

   *reduced_sum = parallel_reduce2(shr_data,threadIdx.x);

}


int main(){
    std::vector<hipComplex> data(NUMELEMENTS,make_hipComplex(1.0,1.0));
    hipComplex sum;

    hipComplex *data_d, *sum_d;

    size_t total_bytes = data.size() * sizeof(hipComplex);

    hipMalloc(&data_d, total_bytes);
    hipMalloc(&sum_d, sizeof(hipComplex));

    hipMemcpy(data_d,data.data(),total_bytes,hipMemcpyHostToDevice);

    dim3 dimGrid(1,1);
    dim3 dimBlock(NUMELEMENTS,1);
    add_numbers<<<dimGrid,dimBlock, total_bytes>>>(data_d,sum_d);

    hipMemcpy(&sum,sum_d,sizeof(hipComplex),hipMemcpyDeviceToHost);

    std::cout << "Resulting sum: (" << sum.x << "," << sum.y << ")" << std::endl;


}