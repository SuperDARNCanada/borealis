#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/complex.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
__global__ void EmptyKernel() {
    //extern __shared__ thrust::complex<float> filter_products[];
}

void throw_on_cuda_error(hipError_t code, const char *file, int line)
{
  if(code != hipSuccess)
  {
    std::stringstream ss;
    ss << file << "(" << line << ")";
    std::string file_and_line;
    ss >> file_and_line;
    throw thrust::system_error(code, thrust::cuda_category(), file_and_line);
  }
}

int main() {

    const int N = 100;

    float time, cumulative_time = 0.f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i=0; i<N; i++) {

        hipEventRecord(start, 0);
        dim3 dimGrid(84,20,1);
        dim3 dimBlock(1024);
        auto bytes = 1024 * sizeof(thrust::complex<float>);
        EmptyKernel<<<dimGrid,dimBlock>>>();
        throw_on_cuda_error(hipPeekAtLastError(), __FILE__,__LINE__);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        cumulative_time = cumulative_time + time;

    }

    printf("Kernel launch overhead time:  %3.5f ms \n", cumulative_time / N);
    return 0;
}