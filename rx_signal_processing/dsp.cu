#include "hip/hip_runtime.h"
/*

Copyright 2017 SuperDARN Canada

See LICENSE for details

  \file dsp.cu
  This file contains the implementation for the all the needed GPU DSP work.
*/

#include "dsp.hpp"

#include <iostream>
#include <cstdlib>
#include <fstream>
#include <string>
#include <sstream>
#include <hip/hip_complex.h>
#include <chrono>
#include <thread>
#include <numeric>
#include <complex>
#include <armadillo>
#include "utils/zmq_borealis_helpers/zmq_borealis_helpers.hpp"
#include "utils/signal_processing_options/signalprocessingoptions.hpp"
#include "utils/protobuf/sigprocpacket.pb.h"
#include "utils/protobuf/processeddata.pb.h"
#include "utils/shared_macros/shared_macros.hpp"
#include "filtering.hpp"
//TODO(keith): decide on handing gpu errors
//TODO(keith): potentially add multigpu support

//This keep postprocess local to this file.
namespace {
  /**
   * @brief      Sends an acknowledgment to the radar control and starts the timing after the
   *             RF samples have been copied.
   *
   * @param[in]  stream           CUDA stream this callback is associated with.
   * @param[in]  status           Error status of CUDA work in the stream.
   * @param[in]  processing_data  A pointer to the DSPCore associated with this CUDA stream.
   */
  void CUDART_CB initial_memcpy_callback_handler(hipStream_t stream, hipError_t status,
                          void *processing_data)
  {
    gpuErrchk(status);

    auto imc = [processing_data]()
    {
      auto dp = static_cast<DSPCore*>(processing_data);
      dp->send_ack();
      dp->start_decimate_timing();
      RUNTIME_MSG(COLOR_RED("Finished initial memcpy handler for sequence #"
                 << dp->get_sequence_num() << ". Thread should exit here"));
    };

    std::thread start_imc(imc);
    start_imc.join();
  }


  /**
   * @brief      Drops samples contaminated by edge effects and filter roll off.
   *
   * @param      input_samples    The input samples.
   * @param      output_samples   The output samples.
   * @param      samps_per_stage  The number of output samples per stage.
   * @param      taps_per_stage   The number of filter taps per stage.
   * @param      dm_rates         The decimation rates of each stage.
   * @param[in]  num_antennas     The number of antennas.
   * @param[in]  num_freqs        The number of freqs.
   */
  void drop_bad_samples(hipComplex *input_samples, std::vector<hipComplex> &output_samples,
                        std::vector<uint32_t> &samps_per_stage,
                        std::vector<uint32_t> &taps_per_stage,
                        std::vector<uint32_t> &dm_rates,
                        uint32_t num_antennas, uint32_t num_freqs)
  {

    auto original_undropped_sample_count = samps_per_stage.back();
    auto original_samples_per_frequency = num_antennas * original_undropped_sample_count;

    // This accounts for the length of the filter extending past the length of input samples while
    // decimating.
    std::vector<uint32_t> bad_samples_per_stage;
    for (uint32_t i=0; i<dm_rates.size(); i++) {
      bad_samples_per_stage.push_back(uint32_t(std::floor(float(taps_per_stage[i]) /
                                                 float(dm_rates[i]))));
    }

    // Propagate the number of bad samples from the first stage through to the last stage.
    for (uint32_t i=1; i<bad_samples_per_stage.size(); i++) {
      bad_samples_per_stage[i] += std::ceil(float(bad_samples_per_stage[i-1])/(dm_rates[i]));
    }

    samps_per_stage.back() -= bad_samples_per_stage.back();
    auto samples_per_frequency = samps_per_stage.back() * num_antennas;

    output_samples.resize(num_freqs * samples_per_frequency);

    for (uint32_t freq_index=0; freq_index < num_freqs; freq_index++) {
      for (int i=0; i<num_antennas; i++){
        auto dest = output_samples.data() + (freq_index * samples_per_frequency) +
                    (i * samps_per_stage.back());
        auto src = input_samples + freq_index * (original_samples_per_frequency) +
                    (i * original_undropped_sample_count);
        auto num_bytes =  sizeof(hipComplex) * samps_per_stage.back();
        memcpy(dest, src, num_bytes);
      }
    }
  }

  /**
   * @brief      Beamforms the final samples
   *
   * @param      filtered_samples         A flat vector containing all the filtered samples for all
   *                                      RX frequencies.
   * @param      beamformed_samples_main  A vector where the beamformed and combined main array
   *                                      samples are placed.
   * @param      beamformed_samples_intf  A vector where the beamformed and combined intf array
   *                                      samples are placed.
   * @param      phases                   A flat vector of the phase delay offsets used to generate
   *                                      azimuthal directions. Phase offsets are complex
   *                                      exponential.
   * @param      num_main_ants            The number of main antennas.
   * @param      num_intf_ants            The number of intf antennas.
   * @param[in]  rx_slice_info            A vector of needed slice metadata.
   * @param      num_samples              The number of samples per antenna.
   *
   *             This method extracts the offsets to the phases and samples needed for the beam
   *             directions of each RX frequency. The Armadillo library is then used to multiply the
   *             matrices to yield the final beamformed samples. The main array and interferometer
   *             array are beamformed separately.
   */
  void beamform_samples(std::vector<hipComplex> &filtered_samples,
                        std::vector<std::vector<hipComplex>> &beamformed_samples_main,
                        std::vector<std::vector<hipComplex>> &beamformed_samples_intf,
                        std::vector<hipComplex> &phases, uint32_t num_main_ants,
                        uint32_t num_intf_ants, std::vector<rx_slice> rx_slice_info,
                        uint32_t num_samples)
  {

    // Gonna make a lambda here to avoid repeated code. This is the main procedure that will
    // beamform the samples from offsets into the vectors.
    auto beamform_from_offsets = [&](hipComplex* samples_ptr,
                                      hipComplex* phases_ptr,
                                      hipComplex* result_ptr,
                                      uint32_t num_antennas, uint32_t num_beams)
    {

      // We work with hipComplex type for most DSP, but Armadillo only knows the equivalent std lib
      // type so we cast to it for this context.
      auto samples_cast = reinterpret_cast<std::complex<float>*>(samples_ptr);
      auto phases_cast = reinterpret_cast<std::complex<float>*>(phases_ptr);

      // All we do here is map an existing set of memory to a structure that Armadillo uses.
      arma::cx_fmat samps(samples_cast, num_samples, num_antennas, false, true);
      arma::cx_fmat phases(phases_cast, num_antennas, num_beams, false, true);

      // Result matrix has dimensions num_samples x num_beams. This means one set of samples for
      // each beam dir. Armadillo overloads the * operator so we dont need to implement any matrix
      // work ourselves.
      arma::cx_fmat result = samps * phases;

      // This piece of code just transforms the Armadillo result back into our flat vector.
      // Armadillo uses column-major ordering, while we use row-major everywhere else. This means
      // that our data will actually be num_beams x num_samps.
      auto beamformed_cast = reinterpret_cast<std::complex<float>*>(result_ptr);
      memcpy(beamformed_cast, result.memptr(), sizeof(std::complex<float>) *
                                        result.n_rows * result.n_cols);
    };

    auto main_phase_offset = 0;

    // Now we calculate the offsets into the samples, phases, and results vector for each
    // RX frequency. Each RX frequency could have a different number of beams, so we increment
    // the phase and results offsets based off the accumulated number of beams. Once we have the
    // offsets, we can call the beamforming lambda.
    for (uint32_t rx_freq_num=0; rx_freq_num<rx_slice_info.size(); rx_freq_num++) {

      auto num_beams = rx_slice_info[rx_freq_num].beam_count;

      // Increment to start of new frequency dataset.
      auto main_sample_offset = num_samples * (num_main_ants + num_intf_ants) * rx_freq_num;
      auto main_sample_ptr = filtered_samples.data() + main_sample_offset;

      auto main_phase_ptr = phases.data() + main_phase_offset;

      auto main_results_ptr = beamformed_samples_main[rx_freq_num].data();

      beamform_from_offsets(main_sample_ptr, main_phase_ptr, main_results_ptr,
                            num_main_ants, num_beams);

      // Only need to worry about beamforming the interferometer if its being used.
      if (num_intf_ants > 0) {

        // Skip the main array samples.
        auto intf_sample_offset = main_sample_offset + (num_samples * num_main_ants);
        auto intf_sample_ptr = filtered_samples.data() + intf_sample_offset;

        auto intf_phase_offset = main_phase_offset + (num_beams * num_main_ants);
        auto intf_phase_ptr = phases.data() + intf_phase_offset;

        // Result offsets will be the same. Each main and intf will have one set of samples for
        // each beam.
        auto intf_results_ptr = beamformed_samples_intf[rx_freq_num].data();

        beamform_from_offsets(intf_sample_ptr, intf_phase_ptr, intf_results_ptr,
                              num_intf_ants, num_beams);
      }

      //Possibly non uniform striding means we incremement the offset as we go.
      main_phase_offset += num_beams * (num_main_ants + num_intf_ants);

    }

  }

  /**
   * @brief      Finds correlations from two sets of samples. Calculates autocorrelation by passing
   *             in the same sample set as both beamformed_samples_1 and beamformed_samples_2.
   *
   * @param      beamformed_samples_1  The first set of beamformed samples for each beam. Both sets
   *                                   of beamformed samples are for a single sequence. The main and
   *                                   intf arrays will have same number of: beams, samples per
   *                                   sequence.
   * @param      beamformed_samples_2  The second set of beamformed samples for each beam.
   * @param      corr_results          A set of vectors where correlation results are stored.
   * @param[in]  rx_slice_info         A vector of the info needed from each slice.
   * @param[in]  num_samples           The number samples for each beam contained in the
   *                                   beamformed_samples set. Assumed to be equal for both sample
   *                                   sets.
   * @param[in]  output_sample_rate    The output sample rate.
   *
   *             For each slice a correlation matrix is build for all the beams in that slice.
   *             Values corresponding to particular lags and range gates are selected from the final
   *             data. This function does not compute the expectation value for the correlations.
   *             That part is done in data write.
   */
  void correlations_from_samples(std::vector<std::vector<hipComplex>> &beamformed_samples_1,
                                  std::vector<std::vector<hipComplex>> &beamformed_samples_2,
                                  std::vector<std::vector<hipComplex>> &corr_results,
                                  std::vector<rx_slice> rx_slice_info, uint32_t num_samples,
                                  double output_sample_rate)
  {
    for (uint32_t slice_num=0; slice_num<rx_slice_info.size(); slice_num++) {
      auto num_beams = rx_slice_info[slice_num].beam_count;
      auto num_ranges = rx_slice_info[slice_num].num_ranges;
      auto num_lags = rx_slice_info[slice_num].lags.size();

      // No need to compute this if there are no lags.
      if (num_lags == 0) {
        continue;
      }

      for (uint32_t beam_count=0; beam_count<num_beams; beam_count++) {
        auto samples_ptr_1 = beamformed_samples_1[slice_num].data();
        auto samples_ptr_2 = beamformed_samples_2[slice_num].data();
        samples_ptr_1 += (beam_count * num_samples);
        samples_ptr_2 += (beam_count * num_samples);

        auto samples_cast_1 = reinterpret_cast<std::complex<float>*>(samples_ptr_1);
        auto samples_cast_2 = reinterpret_cast<std::complex<float>*>(samples_ptr_2);

        // Convert existing memory to Armadillo vectors.
        arma::cx_frowvec samps_1_matrix(samples_cast_1, num_samples, false, true);
        arma::cx_frowvec samps_2_matrix(samples_cast_2, num_samples, false, true);

        // correlation = E(XY^H) where X and Y are random vectors and H is the conjugate.
        // https://en.wikipedia.org/wiki/Autocorrelation_matrix
        // Matrix is not symmetric
        arma::cx_fmat correlation_matrix = samps_1_matrix.t() * samps_2_matrix;

        auto beam_offset = beam_count * num_ranges * num_lags;
        auto first_range_offset = uint32_t(rx_slice_info[slice_num].first_range /
                              rx_slice_info[slice_num].range_sep); // range sep in km, first_range in km
        // Select out the lags for each range gate.
        for(uint32_t range=0; range<num_ranges; range++) {
          for(uint32_t lag=0; lag<num_lags; lag++) {

            // tau spacing is in us, sample rate in hz
            auto tau_in_samples = uint32_t(std::ceil(rx_slice_info[slice_num].tau_spacing * 1e-6 *
                                            output_sample_rate));

            auto p1_offset = rx_slice_info[slice_num].lags[lag].pulse_1 * tau_in_samples;
            auto p2_offset = rx_slice_info[slice_num].lags[lag].pulse_2 * tau_in_samples;

            // use column major indexing.
            auto val = correlation_matrix(range + first_range_offset + p1_offset,
                                          range + first_range_offset + p2_offset);

            auto range_lag_offset = (range * num_lags) + lag;
            auto total_offset = beam_offset + range_lag_offset;
            corr_results[slice_num][total_offset].x = val.real();
            corr_results[slice_num][total_offset].y = val.imag();
          } // close lags scope
        } // close ranges scope
      } // close beams scope
    } // close slices scope
  }

  /**
   * @brief      Creates a data packet of processed data.
   *
   * @param      pd    A processeddata protobuf object.
   * @param      dp    A pointer to the DSPCore object with data to be extracted.
   *
   * This function extracts the processed data into a protobuf that data write can use.
   */
  void create_processed_data_packet(processeddata::ProcessedData &pd, DSPCore* dp)
  {

    std::vector<hipComplex> output_samples;
    auto rx_slice_info = dp->get_slice_info();

    auto samples_per_antenna = dp->get_samples_per_antenna();

    // create a new vector with the number of input rf samples included. Basically the equivalent
    // of a list concat in Python.
    std::vector<uint32_t> samps_per_stage;
    samps_per_stage.push_back(dp->get_num_rf_samples());
    samps_per_stage.insert(samps_per_stage.end(),
                           samples_per_antenna.begin(),
                           samples_per_antenna.end());

    auto filter_taps = dp->get_filter_taps();
    std::vector<uint32_t> taps_per_stage(filter_taps.size());
    for (uint32_t i=0; i<filter_taps.size(); i++) {
      taps_per_stage[i] = filter_taps[i].size();
    }

    auto filter_outputs_h = dp->get_filter_outputs_h();
    auto dm_rates = dp->get_dm_rates();
    drop_bad_samples(filter_outputs_h.back(), output_samples, samps_per_stage, taps_per_stage,
                     dm_rates, dp->get_num_antennas(), rx_slice_info.size());

    // For each antenna, for each frequency.
    auto num_samples_after_dropping = output_samples.size()/
                                      (dp->get_num_antennas()*rx_slice_info.size());





    std::vector<std::vector<hipComplex>> beamformed_samples_main;
    std::vector<std::vector<hipComplex>> beamformed_samples_intf;
    for(auto &rx_slice_info : rx_slice_info) {
      std::vector<hipComplex> main_beam(rx_slice_info.beam_count * num_samples_after_dropping);
      std::vector<hipComplex> intf_beam(rx_slice_info.beam_count * num_samples_after_dropping);
      beamformed_samples_main.push_back(main_beam);
      beamformed_samples_intf.push_back(intf_beam);
    }



    TIMEIT_IF_TRUE_OR_DEBUG(true, "Beamforming time: ",
      {
      auto beam_phases = dp->get_beam_phases();
      beamform_samples(output_samples, beamformed_samples_main, beamformed_samples_intf,
                        beam_phases,
                        dp->sig_options.get_main_antenna_count(),
                        dp->sig_options.get_interferometer_antenna_count(),
                        rx_slice_info,
                        num_samples_after_dropping);
      }
    );

    // set up the vectors ahead of time. Seems to be faster this way.
    std::vector<std::vector<hipComplex>> main_acfs;
    std::vector<std::vector<hipComplex>> xcfs;
    std::vector<std::vector<hipComplex>> intf_acfs;
    for (uint32_t slice_num=0; slice_num<rx_slice_info.size(); slice_num++) {
      auto total_elements = rx_slice_info[slice_num].beam_count *
                            rx_slice_info[slice_num].num_ranges *
                            rx_slice_info[slice_num].lags.size();
      std::vector<hipComplex> v1(total_elements);
      main_acfs.push_back(v1);

      std::vector<hipComplex> v2(total_elements);
      xcfs.push_back(v2);

      std::vector<hipComplex> v3(total_elements);
      intf_acfs.push_back(v3);
    }

    TIMEIT_IF_TRUE_OR_DEBUG(true, "ACF/XCF time: ",
      {
        correlations_from_samples(beamformed_samples_main, beamformed_samples_main,
                                          main_acfs, rx_slice_info,
                                          num_samples_after_dropping, dp->get_output_sample_rate());
        if (dp->sig_options.get_interferometer_antenna_count() > 0) {
          correlations_from_samples(beamformed_samples_main, beamformed_samples_intf,
                                          xcfs, rx_slice_info, num_samples_after_dropping,
                                          dp->get_output_sample_rate());
          correlations_from_samples(beamformed_samples_intf, beamformed_samples_intf,
                                          intf_acfs, rx_slice_info,
                                          num_samples_after_dropping, dp->get_output_sample_rate());
        }
      }

    ); // closing timeit scope

    // We have a lambda to extract the starting pointers of each set of output samples so that
    // we can use a consistent function to write either rf samples or stage data.
    auto make_ptrs_vec = [](hipComplex* output_p, uint32_t num_freqs, uint32_t num_antennas,
                              uint32_t num_samps_per_antenna)
    {
      auto stage_samps_per_set = num_antennas * num_samps_per_antenna;

      std::vector<std::vector<hipComplex*>> ptrs;
      for (uint32_t freq=0; freq<num_freqs; freq++) {
        std::vector<hipComplex*> stage_ptrs;
        for(uint32_t antenna=0; antenna<num_antennas; antenna++) {
          auto idx = freq * stage_samps_per_set + antenna * num_samps_per_antenna;
          stage_ptrs.push_back(output_p + idx);
        }
        ptrs.push_back(stage_ptrs);
      }

      return ptrs;
    };

    std::vector<std::vector<std::vector<hipComplex*>>> all_stage_ptrs;
    #ifdef ENGINEERING_DEBUG
      for (uint32_t i=0; i<filter_outputs_h.size(); i++) {
        auto ptrs = make_ptrs_vec(filter_outputs_h[i], rx_slice_info.size(),
                            dp->get_num_antennas(), samples_per_antenna[i]);
        all_stage_ptrs.push_back(ptrs);
      }
    #endif

    auto output_ptrs = make_ptrs_vec(output_samples.data(), rx_slice_info.size(),
                          dp->get_num_antennas(), num_samples_after_dropping);

    for(uint32_t slice_num=0; slice_num<rx_slice_info.size(); slice_num++) {
      auto dataset = pd.add_outputdataset();
      // This lambda adds the stage data to the processed data for debug purposes.
      auto add_debug_data = [dataset,slice_num](std::string stage_name,
                                                std::vector<hipComplex*> &data_ptrs,
                                                uint32_t num_antennas,
                                                uint32_t num_samps_per_antenna)
      {
        auto debug_samples = dataset->add_debugsamples();

        debug_samples->set_stagename(stage_name);
        for (uint32_t j=0; j<num_antennas; j++){
          auto antenna_data = debug_samples->add_antennadata();
          for(uint32_t k=0; k<num_samps_per_antenna; k++) {
            auto antenna_samp = antenna_data->add_antennasamples();
            antenna_samp->set_real(data_ptrs[j][k].x);
            antenna_samp->set_imag(data_ptrs[j][k].y);
          } // close loop over samples
        } // close loop over antennas
      };

      // Add our beamformed IQ data to the processed data packet that gets sent to data_write.
      for (uint32_t beam_count=0; beam_count<rx_slice_info[slice_num].beam_count; beam_count++) {
        auto beam = dataset->add_beamformedsamples();
        beam->set_beamnum(beam_count);

        for (uint32_t sample=0; sample<num_samples_after_dropping; sample++){
          auto main_sample = beam->add_mainsamples();
          auto beam_start = beam_count * num_samples_after_dropping;
          main_sample->set_real(beamformed_samples_main[slice_num][beam_start + sample].x);
          main_sample->set_imag(beamformed_samples_main[slice_num][beam_start + sample].y);

          if (dp->sig_options.get_interferometer_antenna_count() > 0) {
            auto intf_sample = beam->add_intfsamples();
            intf_sample->set_real(beamformed_samples_intf[slice_num][beam_start + sample].x);
            intf_sample->set_imag(beamformed_samples_intf[slice_num][beam_start + sample].y);
          }
        } // close loop over samples.
      } // close loop over beams.


      auto num_lags = rx_slice_info[slice_num].lags.size();
      auto num_ranges = rx_slice_info[slice_num].num_ranges;
      for (uint32_t beam_count=0; beam_count<rx_slice_info[slice_num].beam_count; beam_count++) {
        auto beam_offset = beam_count * (num_ranges * num_lags);

        for (uint32_t range=0; range<num_ranges; range++) {
          auto range_offset = range * num_lags;

          for (uint32_t lag=0; lag<num_lags; lag++) {
            auto mainacf = dataset->add_mainacf();
            auto val = main_acfs[slice_num][beam_offset + range_offset + lag];
            mainacf->set_real(val.x);
            mainacf->set_imag(val.y);

            if (dp->sig_options.get_interferometer_antenna_count() > 0) {
              auto xcf = dataset->add_xcf();
              auto intfacf = dataset->add_intacf();

              val = xcfs[slice_num][beam_offset + range_offset + lag];
              xcf->set_real(val.x);
              xcf->set_imag(val.y);

              val = intf_acfs[slice_num][beam_offset + range_offset + lag];
              intfacf->set_real(val.x);
              intfacf->set_imag(val.y);
            } // close intf scope
          } // close lag scope
        } // close range scope
      } // close beam scope

      #ifdef ENGINEERING_DEBUG
        for (uint32_t j=0; j<all_stage_ptrs.size(); j++){
          auto stage_str = "stage_" + std::to_string(j);
          add_debug_data(stage_str, all_stage_ptrs[j][slice_num], dp->get_num_antennas(),
            samples_per_antenna[j]);
        }
      #endif

      add_debug_data("antennas", output_ptrs[slice_num], dp->get_num_antennas(),
        num_samples_after_dropping);

      dataset->set_slice_id(rx_slice_info[slice_num].slice_id);
      dataset->set_num_ranges(rx_slice_info[slice_num].num_ranges);
      dataset->set_num_lags(rx_slice_info[slice_num].lags.size());

      DEBUG_MSG("Created dataset for sequence #" << COLOR_RED(dp->get_sequence_num()));
    } // close loop over frequencies (number of slices).

    pd.set_rf_samples_location(dp->get_shared_memory_name());
    pd.set_sequence_num(dp->get_sequence_num());
    pd.set_rx_sample_rate(dp->get_rx_rate());
    pd.set_output_sample_rate(dp->get_output_sample_rate());
    pd.set_processing_time(dp->get_decimate_timing());
    pd.set_initialization_time(dp->get_driver_initialization_time());
    pd.set_sequence_start_time(dp->get_sequence_start_time());
    pd.set_gps_locked(dp->get_gps_locked());
    pd.set_gps_to_system_time_diff(dp->get_gps_to_system_time_diff());
    pd.set_agc_status_bank_h(dp->get_agc_status_bank_h());
    pd.set_lp_status_bank_h(dp->get_lp_status_bank_h());
    pd.set_agc_status_bank_l(dp->get_agc_status_bank_l());
    pd.set_lp_status_bank_l(dp->get_lp_status_bank_l());

  }

  /**
   * @brief      Spawns the postprocessing work after all work in the CUDA stream is completed.
   *
   * @param[in]  stream           CUDA stream this callback is associated with.
   * @param[in]  status           Error status of CUDA work in the stream.
   * @param[in]  processing_data  A pointer to the DSPCore associated with this CUDA stream.
   *
   * The callback itself cannot call anything CUDA related as it may deadlock. It can, however
   * spawn a new thread and then exit gracefully, allowing the thread to do the work.
   */
  void CUDART_CB postprocess(hipStream_t stream, hipError_t status, void *processing_data)
  {

    gpuErrchk(status);

    auto pp = [processing_data]()
    {
      auto dp = static_cast<DSPCore*>(processing_data);

      dp->stop_timing();
      dp->send_timing();


      processeddata::ProcessedData pd;

      TIMEIT_IF_TRUE_OR_DEBUG(true, "Fill + send processed data time ",
        [&]() {
          create_processed_data_packet(pd,dp);
          dp->send_processed_data(pd);
        }()
      );

      RUNTIME_MSG("Cuda kernel timing: " << COLOR_GREEN(dp->get_decimate_timing()) << "ms");
      RUNTIME_MSG("Complete process timing: " << COLOR_GREEN(dp->get_total_timing()) << "ms");
      auto sq_num = dp->get_sequence_num();
      delete dp;

      RUNTIME_MSG(COLOR_RED("Deleted DP in postprocess for sequence #" << sq_num
                  << ". Thread should terminate here."));
    };

    std::thread start_pp(pp);
    start_pp.detach();
  }

}


/**
 * @brief      Gets the properties of each GPU in the system.
 *
 * @return     The gpu properties.
 */
std::vector<hipDeviceProp_t> get_gpu_properties()
{
  std::vector<hipDeviceProp_t> gpu_properties;
  int num_devices = 0;

  gpuErrchk(hipGetDeviceCount(&num_devices));

  for(int i=0; i< num_devices; i++) {
      hipDeviceProp_t properties;
      gpuErrchk(hipGetDeviceProperties(&properties, i));
      gpu_properties.push_back(properties);
  }

  return gpu_properties;
}

/**
 * @brief      Prints the properties of each hipDeviceProp_t in the vector.
 *
 * @param[in]  gpu_properties  A vector of hipDeviceProp_t structs.
 *
 * More info on properties and calculations here:
 * https://devblogs.nvidia.com/parallelforall/how-query-device-properties-and-handle-errors-cuda-cc/
 */
void print_gpu_properties(std::vector<hipDeviceProp_t> gpu_properties) {
  for(auto i : gpu_properties) {
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Device name: " << i.name);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size x: " << i.maxGridSize[0]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size y: " << i.maxGridSize[1]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size z: " << i.maxGridSize[2]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max threads per block: "
                << i.maxThreadsPerBlock);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension x: "
                << i.maxThreadsDim[0]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension y: "
                << i.maxThreadsDim[1]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension z: "
                << i.maxThreadsDim[2]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Memory Clock Rate (GHz): "
                << i.memoryClockRate/1e6);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Memory Bus Width (bits): "
                << i.memoryBusWidth);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Peak Memory Bandwidth (GB/s): "
                << 2.0*i.memoryClockRate*(i.memoryBusWidth/8)/1.0e6);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max shared memory per block: "
                << i.sharedMemPerBlock);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Warpsize: " << i.warpSize);
  }
}


/**
 * @brief      Initializes the parameters needed in order to do asynchronous DSP processing.
 *
 * @param      context                     ZMQ's application context from which to create sockets.
 * @param      sig_options                 The signal processing options.
 * @param[in]  sequence_num                The pulse sequence number for which will be acknowledged.
 * @param[in]  rx_rate                     The USRP sampling rate.
 * @param[in]  output_sample_rate          The final decimated output sample rate.
 * @param[in]  filter_taps                 The filter taps for each stage.
 * @param[in]  beam_phases                 The beam phases.
 * @param[in]  driver_initialization_time  The driver initialization time.
 * @param[in]  sequence_start_time         The sequence start time.
 * @param[in]  gps_locked                  The gps lock status, boolean True if locked.
 * @param[in]  gps_to_system_time_diff     The time diff in seconds btw GPS and system (NTP) time.
 * @param[in]  agc_status_bank_h           The AGC fault status for all TXs on active-high bank
 * @param[in]  lp_status_bank_h            The low power status for all TXs on active-high bank
 * @param[in]  agc_status_bank_l           The AGC fault status for all TXs on active-low bank
 * @param[in]  lp_status_bank_l            The low power status for all TXs on active-low bank
 * @param[in]  dm_rates                    The decimation rates.
 * @param[in]  slice_info                  The slice info given as a vector of rx_slice structs.
 *
 * The constructor creates a new CUDA stream and initializes the timing events. It then opens the
 * shared memory with the received RF samples for a pulse sequence.
 */
DSPCore::DSPCore(zmq::context_t &context, SignalProcessingOptions &sig_options,
                  uint32_t sequence_num, double rx_rate, double output_sample_rate,
                  std::vector<std::vector<float>> filter_taps,
                  std::vector<hipComplex> beam_phases,
                  double driver_initialization_time, double sequence_start_time,
                  bool gps_locked, double gps_to_system_time_diff,
                  uint32_t agc_status_bank_h, uint32_t lp_status_bank_h,
                  uint32_t agc_status_bank_l, uint32_t lp_status_bank_l,
                  std::vector<uint32_t> dm_rates,
                  std::vector<rx_slice> slice_info) :
  sig_options(sig_options),
  sequence_num(sequence_num),
  rx_rate(rx_rate),
  output_sample_rate(output_sample_rate),
  filter_taps(filter_taps),
  beam_phases(beam_phases),
  driver_initialization_time(driver_initialization_time),
  sequence_start_time(sequence_start_time),
  gps_locked(gps_locked),
  gps_to_system_time_diff(gps_to_system_time_diff),
  agc_status_bank_h(agc_status_bank_h),
  lp_status_bank_h(lp_status_bank_h),
  agc_status_bank_l(agc_status_bank_l),
  lp_status_bank_l(lp_status_bank_l),
  dm_rates(dm_rates),
  slice_info(slice_info)
{
  auto identities = {sig_options.get_dspbegin_brian_identity() + std::to_string(sequence_num),
                      sig_options.get_dspend_brian_identity() + std::to_string(sequence_num),
                      sig_options.get_dsp_dw_identity() + std::to_string(sequence_num)};

  zmq_sockets = create_sockets(context, identities, sig_options.get_router_address());

  //https://devblogs.nvidia.com/parallelforall/gpu-pro-tip-cuda-7-streams-simplify-concurrency/
  gpuErrchk(hipStreamCreate(&stream));
  gpuErrchk(hipEventCreate(&initial_start));
  gpuErrchk(hipEventCreate(&kernel_start));
  gpuErrchk(hipEventCreate(&stop));
  gpuErrchk(hipEventCreate(&mem_transfer_end));
  gpuErrchk(hipEventRecord(initial_start, stream));

  shm = SharedMemoryHandler(random_string(20));
}

/**
 * @brief      Frees all associated pointers, events, and streams. Removes and deletes shared
 *             memory.
 */
DSPCore::~DSPCore()
{
  gpuErrchk(hipEventDestroy(initial_start));
  gpuErrchk(hipEventDestroy(kernel_start));
  gpuErrchk(hipEventDestroy(stop));
  gpuErrchk(hipFree(freqs_d));
  gpuErrchk(hipFree(rf_samples_d));
  gpuErrchk(hipFree(bp_filters_d));
  for (auto &filter : lp_filters_d) {
    gpuErrchk(hipFree(filter));
  }

  for (auto &filter_output : filter_outputs_d) {
    gpuErrchk(hipFree(filter_output));
  }

  for (auto &filter_output : filter_outputs_h) {
    gpuErrchk(hipHostFree(filter_output));
  }

  gpuErrchk(hipStreamDestroy(stream));

}

/**
 * @brief      Allocates device memory for the RF samples and then copies them to device.
 *
 * @param[in]  total_antennas         The total number of antennas.
 * @param[in]  num_samples_needed     The number of samples needed from each antenna ringbuffer.
 * @param[in]  extra_samples          The number of extra samples needed for filter propagation.
 * @param[in]  offset_to_first_pulse  Offset from sequence start to center of first pulse.
 * @param[in]  time_zero              The time the driver began collecting samples. seconds since
 *                                    epoch.
 * @param[in]  start_time             The start time of the pulse sequence. seconds since epoch.
 * @param[in]  ringbuffer_size        The ringbuffer size in number of samples.
 * @param      ringbuffer_ptrs_start  A vector of pointers to the start of each antenna ringbuffer.
 *
 * Samples are being stored in a shared memory ringbuffer. This function calculates where to index
 * into the ringbuffer for samples and copies them to the gpu. This function will also copy the
 * samples to a shared memory section that data write, or another process can access in order to
 * work with the raw RF samples.
 */
void DSPCore::allocate_and_copy_rf_samples(uint32_t total_antennas, uint32_t num_samples_needed,
                                int64_t extra_samples, uint32_t offset_to_first_pulse,
                                double time_zero, double start_time,
                                uint64_t ringbuffer_size,
                                std::vector<hipComplex*> &ringbuffer_ptrs_start)
{


  size_t rf_samples_size = total_antennas * num_samples_needed * sizeof(hipComplex);
  shm.create_shr_mem(rf_samples_size);
  gpuErrchk(hipMalloc(&rf_samples_d, rf_samples_size));

  auto sample_time_diff = start_time - time_zero;
  auto sample_in_time = (sample_time_diff * rx_rate) +
                      offset_to_first_pulse -
                      extra_samples;
  auto start_sample = int64_t(std::fmod(sample_in_time, ringbuffer_size));

  if ((start_sample) < 0) {
   start_sample += ringbuffer_size;
  }

  if ((start_sample + num_samples_needed) > ringbuffer_size) {
    for (uint32_t i=0; i<total_antennas; i++) {
      auto first_piece = ringbuffer_size - start_sample;
      auto second_piece = num_samples_needed - first_piece;

      auto first_dest = rf_samples_d + (i*num_samples_needed);
      auto second_dest = rf_samples_d + (i*num_samples_needed) + (first_piece);

      auto first_src = ringbuffer_ptrs_start[i] + start_sample;
      auto second_src = ringbuffer_ptrs_start[i];

      gpuErrchk(hipMemcpyAsync(first_dest, first_src, first_piece * sizeof(hipComplex),
                                 hipMemcpyHostToDevice, stream));
      gpuErrchk(hipMemcpyAsync(second_dest, second_src, second_piece * sizeof(hipComplex),
                                 hipMemcpyHostToDevice, stream));

      auto mem_cast = static_cast<hipComplex*>(shm.get_shrmem_addr());
      auto first_dest_h = mem_cast + (i*num_samples_needed);
      auto second_dest_h = mem_cast + (i*num_samples_needed) + (first_piece);

      memcpy(first_dest_h, first_src, first_piece * sizeof(hipComplex));
      memcpy(second_dest_h, second_src, second_piece * sizeof(hipComplex));
    }

  }
  else {
    for (uint32_t i=0; i<total_antennas; i++) {
      auto dest = rf_samples_d + (i*num_samples_needed);
      auto src = ringbuffer_ptrs_start[i] + start_sample;

      gpuErrchk(hipMemcpyAsync(dest, src, num_samples_needed * sizeof(hipComplex),
        hipMemcpyHostToDevice, stream));

      auto mem_cast = static_cast<hipComplex*>(shm.get_shrmem_addr());
      auto dest_h = mem_cast + (i*num_samples_needed);
      memcpy(dest_h, src, num_samples_needed * sizeof(hipComplex));
    }
  }


}

/**
 * @brief      Allocates device memory for the filtering frequencies and then copies them to device.
 *
 * @param      freqs      A pointer to the filtering freqs.
 * @param[in]  num_freqs  The number of freqs.
 */
void DSPCore::allocate_and_copy_frequencies(void *freqs, uint32_t num_freqs) {
  size_t freqs_size = num_freqs * sizeof(double);
  gpuErrchk(hipMalloc(&freqs_d, freqs_size));
  gpuErrchk(hipMemcpyAsync(freqs_d, freqs, freqs_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocate and copy bandpass filters for all rx freqs to gpu.
 *
 * @param      taps        A pointer to the filter taps.
 * @param[in]  total_taps  The total amount of filter taps.
 */
void DSPCore::allocate_and_copy_bandpass_filters(void *taps, uint32_t total_taps)
{
  size_t bp_filters_size = total_taps * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&bp_filters_d, bp_filters_size));
  gpuErrchk(hipMemcpyAsync(bp_filters_d, taps, bp_filters_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocate and copy a lowpass filter to the gpu.
 *
 * @param      taps        A pointer to the filter taps.
 * @param[in]  total_taps  The total amount of filter taps.
 */
void DSPCore::allocate_and_copy_lowpass_filter(void *taps, uint32_t total_taps)
{
  hipComplex *ptr_d;
  lp_filters_d.push_back(ptr_d);

  size_t filter_size = total_taps * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&lp_filters_d.back(), filter_size));
  gpuErrchk(hipMemcpyAsync(lp_filters_d.back(), taps, filter_size, hipMemcpyHostToDevice, stream));

}

/**
 * @brief      Gets the last filter output d.
 *
 * @return     The last filter output d.
 */
hipComplex* DSPCore::get_last_filter_output_d()
{
  return filter_outputs_d.back();
}

/**
 * @brief      Gets the last pointer stored in the lowpass filters vector.
 *
 * @return     The last lowpass filter pointer inserted into the vector.
 */
hipComplex* DSPCore::get_last_lowpass_filter_d() {
  return lp_filters_d.back();
}

/**
 * @brief      Gets the samples per antenna vector. Vector contains an element for each stage.
 *
 * @return     The samples per antenna vector.
 */
std::vector<uint32_t> DSPCore::get_samples_per_antenna() {
  return samples_per_antenna;
}

/**
 * @brief      The vector containing vectors of filter taps for each stage.
 *
 * @return     The filter taps vectors for each stage.
 */
std::vector<std::vector<float>> DSPCore::get_filter_taps() {
  return filter_taps;
}

/**
 * @brief      Allocate a filter output on the GPU.
 *
 * @param[in]  num_output_samples  The number output samples
 */
void DSPCore::allocate_output(uint32_t num_output_samples)
{
  hipComplex *ptr_d;
  filter_outputs_d.push_back(ptr_d);
  size_t output_size = num_output_samples * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&filter_outputs_d.back(), output_size));

}

/**
 * @brief      Allocate a host pointer for decimation stage output and then copy data.
 *
 * @param[in]  num_output_samples  The number output samples needed.
 * @param      output_d            The device pointer from which to copy from.
 */
void DSPCore::allocate_and_copy_host(uint32_t num_output_samples, hipComplex *output_d)
{
  hipComplex *ptr_h;
  filter_outputs_h.push_back(ptr_h);

  size_t output_size = num_output_samples * sizeof(hipComplex);
  gpuErrchk(hipHostMalloc(&filter_outputs_h.back(), output_size));
  gpuErrchk(hipMemcpyAsync(filter_outputs_h.back(), output_d,
        output_size, hipMemcpyDeviceToHost,stream));

}

/**
 * @brief      Stops the timers that the constructor starts.
 */
void DSPCore::stop_timing()
{
  gpuErrchk(hipEventRecord(stop, stream));
  gpuErrchk(hipEventSynchronize(stop));

  gpuErrchk(hipEventElapsedTime(&total_process_timing_ms, initial_start, stop));
  gpuErrchk(hipEventElapsedTime(&decimate_kernel_timing_ms, kernel_start, stop));
  gpuErrchk(hipEventElapsedTime(&mem_time_ms, initial_start, mem_transfer_end));
  RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Cuda memcpy time for "
    << COLOR_RED("#" << sequence_num) << ": " << COLOR_GREEN(mem_time_ms) << "ms");
  RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Decimate time for "
    << COLOR_RED("#" << sequence_num) << ": "
    << COLOR_GREEN(decimate_kernel_timing_ms) << "ms");

}

/**
 * @brief      Sends the GPU kernel timing to the radar control.
 *
 * The timing here is used as a rate limiter, so that the GPU doesn't become backlogged with data.
 * If the GPU is overburdened, this will result in less averages, but the system wont crash.
 */
void DSPCore::send_timing()
{
  sigprocpacket::SigProcPacket sp;
  sp.set_kerneltime(decimate_kernel_timing_ms);
  sp.set_sequence_num(sequence_num);

  std::string s_msg_str;
  sp.SerializeToString(&s_msg_str);

  auto &timing_socket = zmq_sockets[1];
  auto request = RECV_REQUEST(timing_socket, sig_options.get_brian_dspend_identity());
  SEND_REPLY(timing_socket, sig_options.get_brian_dspend_identity(), s_msg_str);

  DEBUG_MSG(COLOR_RED("Sent timing after processing with sequence #" << sequence_num));

}


/**
 * @brief      Add the postprocessing callback to the stream.
 *
 * This function allocates the host space needed for filter stage data and then copies the data
 * from GPU into the allocated space. Certain DSPCore members needed for post processing are
 * assigned such as the rx freqs, the number of rf samples, the total antennas and the vector
 * of samples per antenna(each stage).
 */
void DSPCore::cuda_postprocessing_callback(uint32_t total_antennas,
                                            uint32_t num_samples_rf,
                                            std::vector<uint32_t> samples_per_antenna,
                                            std::vector<uint32_t> total_output_samples)
{
  #ifdef ENGINEERING_DEBUG
    for (uint32_t i=0; i<filter_outputs_d.size()-1; i++) {
      allocate_and_copy_host(total_output_samples[i], filter_outputs_d[i]);
    }
  #endif

  allocate_and_copy_host(total_output_samples.back(), filter_outputs_d.back());

  num_rf_samples = num_samples_rf;
  num_antennas = total_antennas;
  this->samples_per_antenna = samples_per_antenna;

  gpuErrchk(hipStreamAddCallback(stream, postprocess, this, 0));

  DEBUG_MSG(COLOR_RED("Added stream callback for sequence #" << sequence_num));
}

/**
 * @brief      Sends the acknowledgment to the radar control that the RF samples have been
 *             transfered.
 *
 * RF samples of one pulse sequence can be transfered asynchronously while samples of another are
 * being processed. This means that it is possible to start running a new pulse sequence in the
 * driver as soon as the samples are copied. The asynchronous nature means only timing constraint
 * is the time needed to run the GPU kernels for decimation.
 */
void DSPCore::send_ack()
{
  sigprocpacket::SigProcPacket sp;
  sp.set_sequence_num(sequence_num);

  std::string s_msg_str;
  sp.SerializeToString(&s_msg_str);

  auto &ack_socket = zmq_sockets[0];
  auto request = RECV_REQUEST(ack_socket, sig_options.get_brian_dspbegin_identity());
  SEND_REPLY(ack_socket, sig_options.get_brian_dspbegin_identity(), s_msg_str);

  DEBUG_MSG(COLOR_RED("Sent ack after copy for sequence_num #" << sequence_num));
}

/**
 * @brief      Sends a processed data packet to data write.
 *
 * @param      pd    A processeddata protobuf object.
 */
void DSPCore::send_processed_data(processeddata::ProcessedData &pd)
{
  std::string p_msg_str;
  pd.SerializeToString(&p_msg_str);

  auto &data_socket = zmq_sockets[2];
  SEND_REPLY(data_socket, sig_options.get_dw_dsp_identity(), p_msg_str);

  DEBUG_MSG(COLOR_RED("Send processed data to data_write for sequence #" << sequence_num));
}


/**
 * @brief      Starts the timing before the GPU kernels execute.
 *
 */
void DSPCore::start_decimate_timing()
{
  gpuErrchk(hipEventRecord(kernel_start, stream));
}

/**
 * @brief      Adds the callback to the CUDA stream to acknowledge the RF samples have been copied.
 *
 */
void DSPCore::initial_memcpy_callback()
{
  gpuErrchk(hipEventRecord(mem_transfer_end,stream));
  gpuErrchk(hipStreamAddCallback(stream, initial_memcpy_callback_handler, this, 0));
}


/**
 * @brief      Gets the device pointer to the RF samples.
 *
 * @return     The RF samples device pointer.
 */
hipComplex* DSPCore::get_rf_samples_p(){
  return rf_samples_d;
}

/**
 * @brief      Gets the host pointer to the RF samples.
 *
 * @return     The rf samples host pointer.
 */
std::vector<hipComplex> DSPCore::get_rf_samples_h() {
  return rf_samples_h;
}

/**
 * @brief      Gets the device pointer to the receive frequencies.
 *
 * @return     The frequencies device pointer.
 */
double* DSPCore::get_frequencies_p() {
  return freqs_d;
}


/**
 * @brief      Gets the bandpass filters device pointer.
 *
 * @return     The bandpass filter pointer.
 */
hipComplex* DSPCore::get_bp_filters_p(){
  return bp_filters_d;
}

/**
 * @brief      Gets the vector of decimation rates.
 *
 * @return     The dm rates.
 */
std::vector<uint32_t> DSPCore::get_dm_rates()
{
  return dm_rates;
}

/**
 * @brief      Gets the vector of host side filter outputs.
 *
 * @return     The filter outputs host vector.
 */
std::vector<hipComplex*> DSPCore::get_filter_outputs_h()
{
  return filter_outputs_h;
}

/**
 * @brief      Gets the CUDA stream this DSPCore's work is associated to.
 *
 * @return     The CUDA stream.
 */
hipStream_t DSPCore::get_cuda_stream(){
  return stream;
}

/**
 * @brief      Gets the total GPU process timing in milliseconds.
 *
 * @return     The total process timing.
 */
float DSPCore::get_total_timing()
{
  return total_process_timing_ms;
}

/**
 * @brief      Gets the total decimation timing in milliseconds.
 *
 * @return     The decimation timing.
 */
float DSPCore::get_decimate_timing()
{
  return decimate_kernel_timing_ms;
}

/**
 * @brief      Gets the number of antennas.
 *
 * @return     The number of antennas.
 */
uint32_t DSPCore::get_num_antennas()
{
  return num_antennas;
}

/**
 * @brief      Gets the number of rf samples.
 *
 * @return     The number of rf samples.
 */
uint32_t DSPCore::get_num_rf_samples()
{
  return num_rf_samples;
}

/**
 * @brief      Gets the sequence number.
 *
 * @return     The sequence number.
 */
uint32_t DSPCore::get_sequence_num()
{
  return sequence_num;
}

/**
 * @brief      Gets the rx sample rate.
 *
 * @return     The rx sampling rate (samples per second).
 */
double DSPCore::get_rx_rate()
{
  return rx_rate;
}

/**
 * @brief      Gets the output sample rate.
 *
 * @return     The output decimated and filtered rate (samples per second).
 */
double DSPCore::get_output_sample_rate()
{
  return output_sample_rate;
}

/**
 * @brief     Gets the vector of beam phases.
 *
 * @return    The beam phases.
 */
std::vector<hipComplex> DSPCore::get_beam_phases()
{
  return beam_phases;
}

/**
 * @brief     Gets the name of the shared memory section.
 *
 * @return    The shared memory name string.
 */
std::string DSPCore::get_shared_memory_name()
{
  return shm.get_region_name();
}

/**
 * @brief      Gets the driver initialization timestamp.
 *
 * @return     The driver initialization timestamp.
 */
double DSPCore::get_driver_initialization_time()
{
  return driver_initialization_time;
}

/**
 * @brief      Gets the sequence start timestamp.
 *
 * @return     The sequence start timestamp.
 */
double DSPCore::get_sequence_start_time()
{
  return sequence_start_time;
}

/**
 * @brief      Gets the vector of slice information, rx_slice structs.
 *
 * @return     The vector of rx_slice structs with slice information.
 */
 std::vector<rx_slice> DSPCore::get_slice_info()
{
  return slice_info;
}

/**
 * @brief      Gets the boolean indicating if the GPS is locked or not.
 *
 * @return     The boolean value indicating if the GPS is locked. True if locked.
 */
 bool DSPCore::get_gps_locked()
{
  return gps_locked;
}

/**
 * @brief      Gets the time difference between the GPS (box_time) and system (NTP).
 *
 * @return     The time difference between GPS and system time in seconds. Negative if GPS ahead
 */
double DSPCore::get_gps_to_system_time_diff()
{
  return gps_to_system_time_diff;
}

/**
 * @brief      Gets the AGC fault status word for the TXs connected to the active-high GPIO bank.
 *             A '1' in the bit position indicates an AGC fault.
 *
 * @return     32 bit integer with 1's indicating AGC faults. Each bit position maps to a TX.
 */
uint32_t DSPCore::get_agc_status_bank_h()
{
  return agc_status_bank_h;
}

/**
 * @brief      Gets the low power status word for the TXs connected to the active-high GPIO bank.
 *             A '1' in the bit position indicates a low power condition.
 *
 * @return     32 bit integer with 1's indicating low power conditions. Each bit position maps to a TX.
 */
uint32_t DSPCore::get_lp_status_bank_h()
{
  return lp_status_bank_h;
}

/**
 * @brief      Gets the AGC fault status word for the TXs connected to the active-low GPIO bank.
 *             A '1' in the bit position indicates an AGC fault.
 *
 * @return     32 bit integer with 1's indicating AGC faults. Each bit position maps to a TX.
 */
uint32_t DSPCore::get_agc_status_bank_l()
{
  return agc_status_bank_l;
}

/**
 * @brief      Gets the low power status word for the TXs connected to the active-low GPIO bank.
 *             A '1' in the bit position indicates a low power condition.
 *
 * @return     32 bit integer with 1's indicating low power conditions. Each bit position maps to a TX.
 */
uint32_t DSPCore::get_lp_status_bank_l()
{
  return lp_status_bank_l;
}
