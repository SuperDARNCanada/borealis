#include "hip/hip_runtime.h"
/*

Copyright 2017 SuperDARN Canada

See LICENSE for details

  \file dsp.cu
  This file contains the implementation for the all the needed GPU DSP work.
*/

#include "dsp.hpp"

#include <iostream>
#include <cstdlib>
#include <fstream>
#include <string>
#include <sstream>
#include <hip/hip_complex.h>
#include <chrono>
#include <thread>
#include <complex>
#include <eigen3/Eigen/Dense>
#include "utils/zmq_borealis_helpers/zmq_borealis_helpers.hpp"
#include "utils/signal_processing_options/signalprocessingoptions.hpp"
#include "utils/protobuf/sigprocpacket.pb.h"
#include "utils/protobuf/processeddata.pb.h"
#include "utils/shared_macros/shared_macros.hpp"
#include "filtering.hpp"
//TODO(keith): decide on handing gpu errors
//TODO(keith): potentially add multigpu support

//This keep postprocess local to this file.
namespace {
  /**
   * @brief      Sends an acknowledgment to the radar control and starts the timing after the
   *             RF samples have been copied.
   *
   * @param[in]  stream           CUDA stream this callback is associated with.
   * @param[in]  status           Error status of CUDA work in the stream.
   * @param[in]  processing_data  A pointer to the DSPCore associated with this CUDA stream.
   */
  void CUDART_CB initial_memcpy_callback_handler(hipStream_t stream, hipError_t status,
                          void *processing_data)
  {
    gpuErrchk(status);

    auto imc = [processing_data]()
    {
      auto dp = static_cast<DSPCore*>(processing_data);
      dp->send_ack();
      dp->start_decimate_timing();
      DEBUG_MSG(COLOR_RED("Finished initial memcpy handler for sequence #"
                 << dp->get_sequence_num() << ". Thread should exit here"));
    };

    std::thread start_imc(imc);
    start_imc.join();
  }


  /**
   * @brief      Drops samples contaminated by edge effects and filter roll off.
   *
   * @param      input_samples    The input samples.
   * @param      output_samples   The output samples.
   * @param      samps_per_stage  The number of output samples per stage.
   * @param      taps_per_stage   The number of filter taps per stage.
   * @param[in]  num_antennas     The number of antennas.
   * @param[in]  num_freqs        The number of freqs.
   *
   */
  void drop_bad_samples(hipComplex *input_samples, std::vector<hipComplex> &output_samples,
                        std::vector<uint32_t> &samps_per_stage,
                        std::vector<uint32_t> &taps_per_stage,
                        uint32_t num_antennas, uint32_t num_freqs)
  {
    std::vector<uint32_t> decimation_rates(samps_per_stage.size() - 1);

    for (uint32_t i=0; i<samps_per_stage.size()-1; i++) {
      decimation_rates[i] = samps_per_stage[i]/samps_per_stage[i+1];
    }

    auto original_undropped_sample_count = samps_per_stage.back();
    auto original_samples_per_frequency = num_antennas * original_undropped_sample_count;
    auto num_bad_samples = 0;
    for (int i=0; i<decimation_rates.size(); i++) {
      if (num_bad_samples >= decimation_rates[i]) {
        num_bad_samples = floor(num_bad_samples/decimation_rates[i]);
      }
      else {
        num_bad_samples = 0;
      }

      num_bad_samples += floor(taps_per_stage[i]/decimation_rates[i]);
      if (taps_per_stage[i] % decimation_rates[i] > samps_per_stage[i] % decimation_rates[i]){
          num_bad_samples++;
      }
      samps_per_stage[i+1] -= num_bad_samples;
    }

    auto samples_per_frequency = samps_per_stage.back() * num_antennas;
    output_samples.resize(num_freqs * samples_per_frequency);

    for (uint32_t freq_index=0; freq_index < num_freqs; freq_index++) {
      for (int i=0; i<num_antennas; i++){
        auto dest = output_samples.data() + freq_index*samples_per_frequency + i*samps_per_stage.back();
        auto src = input_samples + freq_index*original_samples_per_frequency + i*original_undropped_sample_count;
        auto num_bytes =  sizeof(hipComplex) * samps_per_stage.back();
        memcpy(dest, src, num_bytes);
      }
    }
  }

  /**
   * @brief      Beamforms the final samples
   *
   * @param      filtered_samples         A flat vector containing all the filtered samples for all
   *                                      RX frequencies.
   * @param      beamformed_samples_main  A vector where the beamformed and combined main array
   *                                      samples are placed.
   * @param      beamformed_samples_intf  A vector where the beamformed and combined intf array
   *                                      samples are placed.
   * @param      phases                   A flat vector of the phase delay offsets used to generate
   *                                      azimuthal directions. Phase offsets are complex
   *                                      exponential.
   * @param      num_main_ants            The number of main antennas.
   * @param      num_intf_ants            The number of intf antennas.
   * @param      beam_direction_counts    A vector containing the number of beam directions for each
   *                                      RX frequency.
   * @param      num_samples              The number of samples per antenna.
   *
   * This method extracts the offsets to the phases and samples needed for the beam directions of
   * each RX frequency. The Eigen library is then used to multiply the matrices to yield the final
   * beamformed samples. The main array and interferometer array are beamformed separately.
   */
  void beamform_samples(std::vector<hipComplex> &filtered_samples,
                        std::vector<hipComplex> &beamformed_samples_main,
                        std::vector<hipComplex> &beamformed_samples_intf,
                        std::vector<hipComplex> &phases, uint32_t num_main_ants,
                        uint32_t num_intf_ants, std::vector<uint32_t> beam_direction_counts,
                        uint32_t num_samples)
  {

    // Gonna make a lambda here to avoid repeated code. This is the main procedure that will
    // beamform the samples from offsets into the vectors.
    auto beamform_from_offsets = [&](hipComplex* samples_ptr,
                                      hipComplex* phases_ptr,
                                      hipComplex* result_ptr,
                                      uint32_t num_antennas, uint32_t num_beams)
    {

      // We work with hipComplex type for most DSP, but Eigen only knows the equivalent std lib type
      // so we cast to it for this context.
      auto samples_cast = reinterpret_cast<std::complex<float>*>(samples_ptr);
      auto phases_cast = reinterpret_cast<std::complex<float>*>(phases_ptr);

      // All we do here is map an existing set of memory to a structure that Eigen uses.
      Eigen::MatrixXcf samps = Eigen::Map<Eigen::Matrix<std::complex<float>,
                                                        Eigen::Dynamic,
                                                        Eigen::Dynamic,
                                                        Eigen::RowMajor>>(samples_cast,
                                                                          num_antennas,
                                                                          num_samples);
      Eigen::MatrixXcf phases = Eigen::Map<Eigen::Matrix<std::complex<float>,
                                                          Eigen::Dynamic,
                                                          Eigen::Dynamic,
                                                          Eigen::RowMajor>>(phases_cast,
                                                                            num_beams,
                                                                            num_antennas);

      // Result matrix has dimensions beams x num_samples. This means one set of samples for
      // each beam dir. Eigen overloads the * operator so we dont need to implement any matrix
      // work ourselves.
      auto result = phases * samps;

      // This piece of code just transforms the Eigen result back into our flat vector.
      auto beamformed_cast = reinterpret_cast<std::complex<float>*>(result_ptr);
      Eigen::Map<Eigen::Matrix<std::complex<float>, Eigen::Dynamic,
                                Eigen::Dynamic, Eigen::RowMajor>>(beamformed_cast, result.rows(),
                                                                  result.cols()) = result;
    };

    auto main_phase_offset = 0;
    auto main_results_offset = 0;

    // Now we calculate the offsets into the samples, phases, and results vector for each
    // RX frequency. Each RX frequency could have a different number of beams, so we increment
    // the phase and results offsets based off the accumulated number of beams. Once we have the
    // offsets, we can call the beamforming lambda.
    for (uint32_t rx_freq_num=0; rx_freq_num<beam_direction_counts.size(); rx_freq_num++) {

      auto num_beams = beam_direction_counts[rx_freq_num];

      // Increment to start of new frequency dataset.
      auto main_sample_offset = num_samples * (num_main_ants + num_intf_ants) * rx_freq_num;
      auto main_sample_ptr = filtered_samples.data() + main_sample_offset;

      auto main_phase_ptr = phases.data() + main_phase_offset;

      auto main_results_ptr = beamformed_samples_main.data() + main_results_offset;

      beamform_from_offsets(main_sample_ptr, main_phase_ptr, main_results_ptr,
                            num_main_ants, num_beams);

      // Only need to worry about beamforming the interferometer if its being used.
      if (num_intf_ants > 0) {

        // Skip the main array samples.
        auto intf_sample_offset = main_sample_offset + (num_samples * num_main_ants);
        auto intf_sample_ptr = filtered_samples.data() + intf_sample_offset;

        auto intf_phase_offset = main_phase_offset + (num_beams * num_main_ants);
        auto intf_phase_ptr = phases.data() + intf_phase_offset;

        // Result offsets will be the same. Each main and intf will have one set of samples for
        // each beam.
        auto intf_results_offset = main_results_offset;
        auto intf_results_ptr = beamformed_samples_intf.data() + intf_results_offset;

        beamform_from_offsets(intf_sample_ptr, intf_phase_ptr, intf_results_ptr,
                              num_intf_ants, num_beams);
      }

      //Possibly non uniform striding means we incremement the offset as we go.
      main_phase_offset += num_beams * (num_main_ants + num_intf_ants);
      main_results_offset += num_beams * num_samples;
    }

  }
  /**
   * @brief      Creates a data packet of processed data.
   *
   * @param      pd    A processeddata protobuf object.
   * @param      dp    A pointer to the DSPCore object with data to be extracted.
   *
   * This function extracts the processed data into a protobuf that data write can use.
   */
  void create_processed_data_packet(processeddata::ProcessedData &pd, DSPCore* dp)
  {

    std::vector<hipComplex> output_samples;

    std::vector<uint32_t> samps_per_stage = {dp->get_num_rf_samples(),
                                             dp->get_num_first_stage_samples_per_antenna(),
                                             dp->get_num_second_stage_samples_per_antenna(),
                                             dp->get_num_third_stage_samples_per_antenna(),
                                             dp->get_num_fourth_stage_samples_per_antenna()};
    std::vector<uint32_t> taps_per_stage = {dp->dsp_filters->get_num_first_stage_taps(),
                                            dp->dsp_filters->get_num_second_stage_taps(),
                                            dp->dsp_filters->get_num_third_stage_taps(),
                                            dp->dsp_filters->get_num_fourth_stage_taps()};

    drop_bad_samples(dp->get_host_output_h(), output_samples, samps_per_stage, taps_per_stage,
                     dp->get_num_antennas(), dp->get_rx_freqs().size());

    // For each antenna, for each frequency.
    auto num_samples_after_dropping = output_samples.size()/
                                      (dp->get_num_antennas()*dp->get_rx_freqs().size());




    auto total_beam_dirs = 0;
    auto beam_direction_counts = dp->get_beam_direction_counts();
    for(auto &beam_count : beam_direction_counts) {
      total_beam_dirs += beam_count;
    }

    std::vector<hipComplex> beamformed_samples_main(total_beam_dirs * num_samples_after_dropping);
    std::vector<hipComplex> beamformed_samples_intf(total_beam_dirs * num_samples_after_dropping);

    TIMEIT_IF_TRUE_OR_DEBUG(false,"Beamforming time: ",
      {
      auto beam_phases = dp->get_beam_phases();
      beamform_samples(output_samples, beamformed_samples_main, beamformed_samples_intf,
                        beam_phases,
                        dp->sig_options.get_main_antenna_count(),
                        dp->sig_options.get_interferometer_antenna_count(),
                        beam_direction_counts,
                        num_samples_after_dropping);
      }
    );

    // We have a lambda to extract the starting pointers of each set of output samples so that
    // we can use a consistent function to write either rf samples or stage data.
    auto make_ptrs_vec = [](hipComplex* output_p, uint32_t num_freqs, uint32_t num_antennas,
                              uint32_t num_samps_per_antenna)
    {
      auto stage_samps_per_set = num_antennas * num_samps_per_antenna;

      std::vector<std::vector<hipComplex*>> ptrs;
      for (uint32_t freq=0; freq<num_freqs; freq++) {
        std::vector<hipComplex*> stage_ptrs;
        for(uint32_t antenna=0; antenna<num_antennas; antenna++) {
          auto idx = freq * stage_samps_per_set + antenna * num_samps_per_antenna;
          stage_ptrs.push_back(output_p + idx);
        }
        ptrs.push_back(stage_ptrs);
      }

      return ptrs;
    };

    #ifdef ENGINEERING_DEBUG
      auto stage_1_ptrs = make_ptrs_vec(dp->get_first_stage_output_h(), dp->get_rx_freqs().size(),
                            dp->get_num_antennas(),dp->get_num_first_stage_samples_per_antenna());

      auto stage_2_ptrs = make_ptrs_vec(dp->get_second_stage_output_h(), dp->get_rx_freqs().size(),
                            dp->get_num_antennas(),dp->get_num_second_stage_samples_per_antenna());

      auto stage_3_ptrs = make_ptrs_vec(dp->get_third_stage_output_h(), dp->get_rx_freqs().size(),
                              dp->get_num_antennas(),dp->get_num_third_stage_samples_per_antenna());

      auto stage_4_ptrs = make_ptrs_vec(dp->get_fourth_stage_output_h(), dp->get_rx_freqs().size(),
                              dp->get_num_antennas(),dp->get_num_fourth_stage_samples_per_antenna());
    #endif

    auto output_ptrs = make_ptrs_vec(output_samples.data(), dp->get_rx_freqs().size(),
                          dp->get_num_antennas(), num_samples_after_dropping);

    auto beamformed_offset = 0;
    for(uint32_t i=0; i<dp->get_rx_freqs().size(); i++) {
      auto dataset = pd.add_outputdataset();
      // This lambda adds the stage data to the processed data for debug purposes.
      auto add_debug_data = [dataset,i](std::string stage_name, std::vector<hipComplex*> &data_ptrs,
                                          uint32_t num_antennas, uint32_t num_samps_per_antenna)
      {
        auto debug_samples = dataset->add_debugsamples();

        debug_samples->set_stagename(stage_name);
        for (uint32_t j=0; j<num_antennas; j++){
          auto antenna_data = debug_samples->add_antennadata();
          for(uint32_t k=0; k<num_samps_per_antenna; k++) {
            auto antenna_samp = antenna_data->add_antennasamples();
            antenna_samp->set_real(data_ptrs[j][k].x);
            antenna_samp->set_imag(data_ptrs[j][k].y);
          } // close loop over samples
        } // close loop over antennas
      };

      // Add our beamformed IQ data to the processed data packet that gets sent to data_write.
      for (uint32_t beam_count=0; beam_count<beam_direction_counts[i]; beam_count++) {
        auto beam = dataset->add_beamformedsamples();
        beam->set_beamnum(beam_count);

        for (uint32_t sample=0; sample<num_samples_after_dropping; sample++){
          auto main_sample = beam->add_mainsamples();
          main_sample->set_real(beamformed_samples_main[beamformed_offset + sample].x);
          main_sample->set_imag(beamformed_samples_main[beamformed_offset + sample].y);

          if (dp->sig_options.get_interferometer_antenna_count() > 0) {
            auto intf_sample = beam->add_intfsamples();
            intf_sample->set_real(beamformed_samples_intf[beamformed_offset + sample].x);
            intf_sample->set_imag(beamformed_samples_intf[beamformed_offset + sample].y);
          }
        } // close loop over samples.
      } // close loop over beams.

      // Keep track of offsets as we move along frequencies. Different frequencies can have
      // different beams.
      beamformed_offset += beam_direction_counts[i];


      #ifdef ENGINEERING_DEBUG
        add_debug_data("stage_1",stage_1_ptrs[i],dp->get_num_antennas(),
                    dp->get_num_first_stage_samples_per_antenna());
        add_debug_data("stage_2",stage_2_ptrs[i],dp->get_num_antennas(),
                    dp->get_num_second_stage_samples_per_antenna());
        add_debug_data("stage_3",stage_3_ptrs[i],dp->get_num_antennas(),
                    dp->get_num_third_stage_samples_per_antenna());
        add_debug_data("stage_4",stage_4_ptrs[i],dp->get_num_antennas(),
            dp->get_num_fourth_stage_samples_per_antenna());
      #endif

      add_debug_data("output_samples", output_ptrs[i], dp->get_num_antennas(),
        num_samples_after_dropping);

      dataset->set_slice_id(dp->get_slice_ids()[i]);
      DEBUG_MSG("Created dataset for sequence #" << COLOR_RED(dp->get_sequence_num()));
    } // close loop over frequencies.

    pd.set_rf_samples_location(dp->get_shared_memory_name());
    pd.set_sequence_num(dp->get_sequence_num());
    pd.set_rx_sample_rate(dp->get_rx_rate());
    pd.set_output_sample_rate(dp->get_output_sample_rate());
    pd.set_processing_time(dp->get_decimate_timing());
    pd.set_initialization_time(dp->get_driver_initialization_time());
    pd.set_sequence_start_time(dp->get_sequence_start_time());
  }

  /**
   * @brief      Spawns the postprocessing work after all work in the CUDA stream is completed.
   *
   * @param[in]  stream           CUDA stream this callback is associated with.
   * @param[in]  status           Error status of CUDA work in the stream.
   * @param[in]  processing_data  A pointer to the DSPCore associated with this CUDA stream.
   *
   * The callback itself cannot call anything CUDA related as it may deadlock. It can, however
   * spawn a new thread and then exit gracefully, allowing the thread to do the work.
   */
  void CUDART_CB postprocess(hipStream_t stream, hipError_t status, void *processing_data)
  {
    gpuErrchk(status);

    auto pp = [processing_data]()
    {
      auto dp = static_cast<DSPCore*>(processing_data);

      dp->stop_timing();
      dp->send_timing();


      processeddata::ProcessedData pd;

      TIMEIT_IF_TRUE_OR_DEBUG(false, "Fill + send processed data time ",
        [&]() {
          create_processed_data_packet(pd,dp);
          dp->send_processed_data(pd);
        }()
      );

      DEBUG_MSG("Cuda kernel timing: " << COLOR_GREEN(dp->get_decimate_timing()) << "ms");
      DEBUG_MSG("Complete process timing: " << COLOR_GREEN(dp->get_total_timing()) << "ms");
      auto sq_num = dp->get_sequence_num();
      delete dp;

      DEBUG_MSG(COLOR_RED("Deleted DP in postprocess for sequence #" << sq_num
                  << ". Thread should terminate here."));
    };

    std::thread start_pp(pp);
    start_pp.detach();
  }

}


/**
 * @brief      Gets the properties of each GPU in the system.
 *
 * @return     The gpu properties.
 */
std::vector<hipDeviceProp_t> get_gpu_properties()
{
  std::vector<hipDeviceProp_t> gpu_properties;
  int num_devices = 0;

  gpuErrchk(hipGetDeviceCount(&num_devices));

  for(int i=0; i< num_devices; i++) {
      hipDeviceProp_t properties;
      gpuErrchk(hipGetDeviceProperties(&properties, i));
      gpu_properties.push_back(properties);
  }

  return gpu_properties;
}

/**
 * @brief      Prints the properties of each hipDeviceProp_t in the vector.
 *
 * @param[in]  gpu_properties  A vector of hipDeviceProp_t structs.
 *
 * More info on properties and calculations here:
 * https://devblogs.nvidia.com/parallelforall/how-query-device-properties-and-handle-errors-cuda-cc/
 */
void print_gpu_properties(std::vector<hipDeviceProp_t> gpu_properties) {
  for(auto i : gpu_properties) {
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Device name: " << i.name);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size x: " << i.maxGridSize[0]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size y: " << i.maxGridSize[1]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max grid size z: " << i.maxGridSize[2]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max threads per block: "
                << i.maxThreadsPerBlock);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension x: "
                << i.maxThreadsDim[0]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension y: "
                << i.maxThreadsDim[1]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max size of block dimension z: "
                << i.maxThreadsDim[2]);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Memory Clock Rate (GHz): "
                << i.memoryClockRate/1e6);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Memory Bus Width (bits): "
                << i.memoryBusWidth);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Peak Memory Bandwidth (GB/s): "
                << 2.0*i.memoryClockRate*(i.memoryBusWidth/8)/1.0e6);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Max shared memory per block: "
                << i.sharedMemPerBlock);
    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "  Warpsize: " << i.warpSize);
  }
}


///TODO(keith): update docstring
/**
 * @brief      Initializes the parameters needed in order to do asynchronous DSP processing.
 *
 * @param      ack_s         A pointer to the socket used for acknowledging when the transfer of RF
 *                           samples has completed.
 * @param[in]  timing_s      A pointer to the socket used for reporting GPU kernel timing.
 * @param[in]  sq_num        The pulse sequence number for which will be acknowledged.
 * @param[in]  shr_mem_name  The char string used to open a section of shared memory with RF
 *                           samples.
 *
 * The constructor creates a new CUDA stream and initializes the timing events. It then opens
 * the shared memory with the received RF samples for a pulse sequence.
 */
DSPCore::DSPCore(zmq::socket_t *ack_socket, zmq::socket_t *timing_socket, zmq::socket_t *data_socket,
                  SignalProcessingOptions &sig_options, uint32_t sequence_num,
                  double rx_rate, double output_sample_rate, std::vector<double> rx_freqs, Filtering *dsp_filters,
                  std::vector<hipComplex> beam_phases, std::vector<uint32_t> beam_direction_counts,
                  double driver_initialization_time, double sequence_start_time,
                  std::vector<uint32_t> slice_ids) :
  sequence_num(sequence_num),
  rx_rate(rx_rate),
  output_sample_rate(output_sample_rate),
  ack_socket(ack_socket),
  timing_socket(timing_socket),
  data_socket(data_socket),
  rx_freqs(rx_freqs),
  sig_options(sig_options),
  dsp_filters(dsp_filters),
  beam_phases(beam_phases),
  beam_direction_counts(beam_direction_counts),
  driver_initialization_time(driver_initialization_time),
  sequence_start_time(sequence_start_time),
  slice_ids(slice_ids)
{

  //https://devblogs.nvidia.com/parallelforall/gpu-pro-tip-cuda-7-streams-simplify-concurrency/
  gpuErrchk(hipStreamCreate(&stream));
  gpuErrchk(hipEventCreate(&initial_start));
  gpuErrchk(hipEventCreate(&kernel_start));
  gpuErrchk(hipEventCreate(&stop));
  gpuErrchk(hipEventCreate(&mem_transfer_end));
  gpuErrchk(hipEventRecord(initial_start, stream));

  shm = SharedMemoryHandler(random_string(20));


}

/**
 * @brief      Frees all associated pointers, events, and streams. Removes and deletes shared
 *             memory.
 */
DSPCore::~DSPCore()
{
  gpuErrchk(hipEventDestroy(initial_start));
  gpuErrchk(hipEventDestroy(kernel_start));
  gpuErrchk(hipEventDestroy(stop));
  gpuErrchk(hipFree(freqs_d));
  gpuErrchk(hipFree(rf_samples_d));
  gpuErrchk(hipFree(first_stage_bp_filters_d));
  gpuErrchk(hipFree(second_stage_filter_d));
  gpuErrchk(hipFree(third_stage_filter_d));
  gpuErrchk(hipFree(fourth_stage_filter_d));
  gpuErrchk(hipFree(first_stage_output_d));
  gpuErrchk(hipFree(second_stage_output_d));
  gpuErrchk(hipFree(third_stage_output_d));
  gpuErrchk(hipFree(fourth_stage_output_d));
  #ifdef ENGINEERING_DEBUG
    gpuErrchk(hipHostFree(first_stage_output_h));
    gpuErrchk(hipHostFree(second_stage_output_h));
    gpuErrchk(hipHostFree(third_stage_output_h));
    gpuErrchk(hipHostFree(fourth_stage_output_h));
  #endif
  gpuErrchk(hipHostFree(host_output_h));
  gpuErrchk(hipStreamDestroy(stream));

}

/**
 * @brief      Allocates device memory for the RF samples and then copies them to device.
 *
 * @param[in]  total_antennas         The total number of antennas.
 * @param[in]  num_samples_needed     The number of samples needed from each antenna ringbuffer.
 * @param[in]  extra_samples          The number of extra samples needed for filter propagation.
 * @param[in]  offset_to_first_pulse  Offset from sequence start to center of first pulse.
 * @param[in]  time_zero              The time the driver began collecting samples. seconds since
 *                                    epoch.
 * @param[in]  start_time             The start time of the pulse sequence. seconds since epoch.
 * @param[in]  ringbuffer_size        The ringbuffer size in number of samples.
 * @param      ringbuffer_ptrs_start  A vector of pointers to the start of each antenna ringbuffer.
 *
 * Samples are being stored in a shared memory ringbuffer. This function calculates where to index
 * into the ringbuffer for samples and copies them to the gpu. This function will also copy the
 * samples to a shared memory section that data write, or another process can access in order to
 * work with the raw RF samples.
 */
void DSPCore::allocate_and_copy_rf_samples(uint32_t total_antennas, uint32_t num_samples_needed,
                                int64_t extra_samples, uint32_t offset_to_first_pulse,
                                double time_zero, double start_time,
                                uint64_t ringbuffer_size,
                                std::vector<hipComplex*> &ringbuffer_ptrs_start)
{


  size_t rf_samples_size = total_antennas * num_samples_needed * sizeof(hipComplex);
  shm.create_shr_mem(rf_samples_size);
  gpuErrchk(hipMalloc(&rf_samples_d, rf_samples_size));

  auto sample_time_diff = start_time - time_zero;
  auto sample_in_time = (sample_time_diff * rx_rate) +
                      offset_to_first_pulse -
                      extra_samples;
  auto start_sample = int64_t(std::fmod(sample_in_time, ringbuffer_size));

  if ((start_sample) < 0) {
   start_sample += ringbuffer_size;
  }

  if ((start_sample + num_samples_needed) > ringbuffer_size) {
    for (uint32_t i=0; i<total_antennas; i++) {
      auto first_piece = ringbuffer_size - start_sample;
      auto second_piece = num_samples_needed - first_piece;

      auto first_dest = rf_samples_d + (i*num_samples_needed);
      auto second_dest = rf_samples_d + (i*num_samples_needed) + (first_piece);

      auto first_src = ringbuffer_ptrs_start[i] + start_sample;
      auto second_src = ringbuffer_ptrs_start[i];

      gpuErrchk(hipMemcpyAsync(first_dest, first_src, first_piece * sizeof(hipComplex),
                                 hipMemcpyHostToDevice, stream));
      gpuErrchk(hipMemcpyAsync(second_dest, second_src, second_piece * sizeof(hipComplex),
                                 hipMemcpyHostToDevice, stream));

      auto mem_cast = static_cast<hipComplex*>(shm.get_shrmem_addr());
      auto first_dest_h = mem_cast + (i*num_samples_needed);
      auto second_dest_h = mem_cast + (i*num_samples_needed) + (first_piece);

      memcpy(first_dest_h, first_src, first_piece * sizeof(hipComplex));
      memcpy(second_dest_h, second_src, second_piece * sizeof(hipComplex));
    }

  }
  else {
    for (uint32_t i=0; i<total_antennas; i++) {
      auto dest = rf_samples_d + (i*num_samples_needed);
      auto src = ringbuffer_ptrs_start[i] + start_sample;

      gpuErrchk(hipMemcpyAsync(dest, src, num_samples_needed * sizeof(hipComplex),
        hipMemcpyHostToDevice, stream));

      auto mem_cast = static_cast<hipComplex*>(shm.get_shrmem_addr());
      auto dest_h = mem_cast + (i*num_samples_needed);
      memcpy(dest_h, src, num_samples_needed * sizeof(hipComplex));
    }
  }


}

/**
 * @brief      Allocates device memory for the filtering frequencies and then copies them to device.
 *
 * @param      freqs      A pointer to the filtering freqs.
 * @param[in]  num_freqs  The number of freqs.
 */
void DSPCore::allocate_and_copy_frequencies(void *freqs, uint32_t num_freqs) {
  size_t freqs_size = num_freqs * sizeof(double);
  gpuErrchk(hipMalloc(&freqs_d, freqs_size));
  gpuErrchk(hipMemcpyAsync(freqs_d, freqs, freqs_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocates device memory for the first stage filters and then copies them to the
 *             device.
 *
 * @param[in]  taps        A pointer to the first stage filter taps.
 * @param[in]  total_taps  The total number of taps for all filters.
 */
void DSPCore::allocate_and_copy_first_stage_filters(void *taps, uint32_t total_taps)
{
  size_t first_stage_bp_filters_size = total_taps * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&first_stage_bp_filters_d, first_stage_bp_filters_size));
  gpuErrchk(hipMemcpyAsync(first_stage_bp_filters_d, taps,
        first_stage_bp_filters_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocates device memory for the second stage filter and then copies it to the
 *             device.
 *
 * @param[in]  taps        A pointer to the second stage filter taps.
 * @param[in]  total_taps  The total number of taps for all filters.
 */
void DSPCore::allocate_and_copy_second_stage_filter(void *taps, uint32_t total_taps)
{
  size_t second_stage_filter_size = total_taps * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&second_stage_filter_d, second_stage_filter_size));
  gpuErrchk(hipMemcpyAsync(second_stage_filter_d, taps,
         second_stage_filter_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocates device memory for the third stage filter and then copies it to the
 *             device.
 *
 * @param[in]  taps        A pointer to the third stage filters.
 * @param[in]  total_taps  The total number of taps for all filters.
 */
void DSPCore::allocate_and_copy_third_stage_filter(void *taps, uint32_t total_taps)
{
  size_t third_stage_filter_size = total_taps * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&third_stage_filter_d, third_stage_filter_size));
  gpuErrchk(hipMemcpyAsync(third_stage_filter_d, taps,
        third_stage_filter_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocates device memory for the fourth stage filter and then copies it to the
 *             device.
 *
 * @param[in]  taps        A pointer to the fourth stage filters.
 * @param[in]  total_taps  The total number of taps for all filters.
 */
void DSPCore::allocate_and_copy_fourth_stage_filter(void *taps, uint32_t total_taps)
{
  size_t fourth_stage_filter_size = total_taps * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&fourth_stage_filter_d, fourth_stage_filter_size));
  gpuErrchk(hipMemcpyAsync(fourth_stage_filter_d, taps,
        fourth_stage_filter_size, hipMemcpyHostToDevice, stream));
}

/**
 * @brief      Allocates device memory for the output of the first stage filters.
 *
 * @param[in]  num_first_stage_output_samples  The total number of output samples from first
 *                                             stage.
 */
void DSPCore::allocate_first_stage_output(uint32_t num_first_stage_output_samples)
{
  size_t first_stage_output_size = num_first_stage_output_samples * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&first_stage_output_d, first_stage_output_size));
}

/**
 * @brief      Allocates device memory for the output of the second stage filters.
 *
 * @param[in]  num_second_stage_output_samples  The total number of output samples from second
 *             stage.
 */
void DSPCore::allocate_second_stage_output(uint32_t num_second_stage_output_samples)
{
  size_t second_stage_output_size = num_second_stage_output_samples * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&second_stage_output_d, second_stage_output_size));
}

/**
 * @brief      Allocates device memory for the output of the third stage filters.
 *
 * @param[in]  num_third_stage_output_samples  The total number of output samples from third
 *                                             stage.
 */
void DSPCore::allocate_third_stage_output(uint32_t num_third_stage_output_samples)
{
  size_t third_stage_output_size = num_third_stage_output_samples * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&third_stage_output_d, third_stage_output_size));
}

/**
 * @brief      Allocates device memory for the output of the fourth stage filters.
 *
 * @param[in]  num_fourth_stage_output_samples The total number of output samples from fourth
 *                                             stage.
 */
void DSPCore::allocate_fourth_stage_output(uint32_t num_fourth_stage_output_samples)
{
  size_t fourth_stage_output_size = num_fourth_stage_output_samples * sizeof(hipComplex);
  gpuErrchk(hipMalloc(&fourth_stage_output_d, fourth_stage_output_size));
}

/**
 * @brief      Allocates host memory for final decimated samples and copies from device to host.
 *
 * @param[in]  num_host_samples  Number of host samples to copy back from device.
 */
void DSPCore::allocate_and_copy_host_output(uint32_t num_host_samples)
{
  size_t host_output_size = num_host_samples * sizeof(hipComplex);
  gpuErrchk(hipHostAlloc(&host_output_h, host_output_size, hipHostMallocDefault));
  gpuErrchk(hipMemcpyAsync(host_output_h, fourth_stage_output_d,
        host_output_size, hipMemcpyDeviceToHost,stream));
}


/**
 * @brief      Allocates host memory for the first stage samples and copies from device to host.
 *
 * @param[in]  num_first_stage_output_samples  The number of first stage output samples.
 */
void DSPCore::allocate_and_copy_first_stage_host(uint32_t num_first_stage_output_samples)
{
  size_t host_output_size = num_first_stage_output_samples * sizeof(hipComplex);
  gpuErrchk(hipHostAlloc(&first_stage_output_h, host_output_size, hipHostMallocDefault));
  gpuErrchk(hipMemcpyAsync(first_stage_output_h, first_stage_output_d,
        host_output_size, hipMemcpyDeviceToHost,stream));
}

/**
 * @brief      Allocates host memory for the second stage samples and copies from device to host.
 *
 * @param[in]  num_second_stage_output_samples  The number of second stage output samples.
 */
void DSPCore::allocate_and_copy_second_stage_host(uint32_t num_second_stage_output_samples)
{
  size_t host_output_size = num_second_stage_output_samples * sizeof(hipComplex);
  gpuErrchk(hipHostAlloc(&second_stage_output_h, host_output_size, hipHostMallocDefault));
  gpuErrchk(hipMemcpyAsync(second_stage_output_h, second_stage_output_d,
        host_output_size, hipMemcpyDeviceToHost,stream));
}

/**
 * @brief      Allocates host memory for the third stage samples and copies from device to host.
 *
 * @param[in]  num_third_stage_output_samples  The number of third stage output samples.
 */
void DSPCore::allocate_and_copy_third_stage_host(uint32_t num_third_stage_output_samples)
{
  size_t host_output_size = num_third_stage_output_samples * sizeof(hipComplex);
  gpuErrchk(hipHostAlloc(&third_stage_output_h, host_output_size, hipHostMallocDefault));
  gpuErrchk(hipMemcpyAsync(third_stage_output_h, third_stage_output_d,
        host_output_size, hipMemcpyDeviceToHost,stream));
}

/**
 * @brief      Allocates host memory for the fourth stage samples and copies from device to host.
 *
 * @param[in]  num_fourth_stage_output_samples  The number of fourth stage output samples.
 */
void DSPCore::allocate_and_copy_fourth_stage_host(uint32_t num_fourth_stage_output_samples)
{
  size_t host_output_size = num_fourth_stage_output_samples * sizeof(hipComplex);
  gpuErrchk(hipHostAlloc(&fourth_stage_output_h, host_output_size, hipHostMallocDefault));
  gpuErrchk(hipMemcpyAsync(fourth_stage_output_h, fourth_stage_output_d,
        host_output_size, hipMemcpyDeviceToHost,stream));
}

/**
 * @brief      Stops the timers that the constructor starts.
 */
void DSPCore::stop_timing()
{
  gpuErrchk(hipEventRecord(stop, stream));
  gpuErrchk(hipEventSynchronize(stop));

  gpuErrchk(hipEventElapsedTime(&total_process_timing_ms, initial_start, stop));
  gpuErrchk(hipEventElapsedTime(&decimate_kernel_timing_ms, kernel_start, stop));
  gpuErrchk(hipEventElapsedTime(&mem_time_ms, initial_start, mem_transfer_end));
  RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Cuda memcpy time: "
    << COLOR_GREEN(mem_time_ms) << "ms");
  RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Decimate time: "
    << COLOR_GREEN(decimate_kernel_timing_ms) << "ms");

}

/**
 * @brief      Sends the GPU kernel timing to the radar control.
 *
 * The timing here is used as a rate limiter, so that the GPU doesn't become backlogged with data.
 * If the GPU is overburdened, this will result in less averages, but the system wont crash.
 */
void DSPCore::send_timing()
{
  sigprocpacket::SigProcPacket sp;
  sp.set_kerneltime(decimate_kernel_timing_ms);
  sp.set_sequence_num(sequence_num);

  std::string s_msg_str;
  sp.SerializeToString(&s_msg_str);

  auto request = RECV_REQUEST(*timing_socket, sig_options.get_brian_dspend_identity());
  SEND_REPLY(*timing_socket, sig_options.get_brian_dspend_identity(), s_msg_str);

  DEBUG_MSG(COLOR_RED("Sent timing after processing with sequence #" << sequence_num));

}


/**
 * @brief      Add the postprocessing callback to the stream.
 *
 */
void DSPCore::cuda_postprocessing_callback(std::vector<double> freqs, uint32_t total_antennas,
                                            uint32_t num_samples_rf,
                                            uint32_t num_output_samples_per_antenna_1,
                                            uint32_t num_output_samples_per_antenna_2,
                                            uint32_t num_output_samples_per_antenna_3,
                                            uint32_t num_output_samples_per_antenna_4)
{
    #ifdef ENGINEERING_DEBUG
      auto total_output_samples_1 = num_output_samples_per_antenna_1 * rx_freqs.size() *
                                      total_antennas;
      auto total_output_samples_2 = num_output_samples_per_antenna_2 * rx_freqs.size() *
                                      total_antennas;
      auto total_output_samples_3 = num_output_samples_per_antenna_3 * rx_freqs.size() *
                                      total_antennas;
      auto total_output_samples_4 = num_output_samples_per_antenna_4 * rx_freqs.size() *
                                      total_antennas;

      allocate_and_copy_first_stage_host(total_output_samples_1);
      allocate_and_copy_second_stage_host(total_output_samples_2);
      allocate_and_copy_third_stage_host(total_output_samples_3);
      allocate_and_copy_fourth_stage_host(total_output_samples_4);

    #endif

    rx_freqs = freqs;
    num_rf_samples = num_samples_rf;
    num_antennas = total_antennas;
    num_first_stage_samples_per_antenna = num_output_samples_per_antenna_1;
    num_second_stage_samples_per_antenna = num_output_samples_per_antenna_2;
    num_third_stage_samples_per_antenna = num_output_samples_per_antenna_3;
    num_fourth_stage_samples_per_antenna = num_output_samples_per_antenna_4;

    gpuErrchk(hipStreamAddCallback(stream, postprocess, this, 0));

    DEBUG_MSG(COLOR_RED("Added stream callback for sequence #" << sequence_num));
}

/**
 * @brief      Sends the acknowledgment to the radar control that the RF samples have been
 *             transfered.
 *
 * RF samples of one pulse sequence can be transfered asynchronously while samples of another are
 * being processed. This means that it is possible to start running a new pulse sequence in the
 * driver as soon as the samples are copied. The asynchronous nature means only timing constraint
 * is the time needed to run the GPU kernels for decimation.
 */
void DSPCore::send_ack()
{
  sigprocpacket::SigProcPacket sp;
  sp.set_sequence_num(sequence_num);

  std::string s_msg_str;
  sp.SerializeToString(&s_msg_str);

  auto request = RECV_REQUEST(*ack_socket, sig_options.get_brian_dspbegin_identity());
  SEND_REPLY(*ack_socket, sig_options.get_brian_dspbegin_identity(), s_msg_str);

  DEBUG_MSG(COLOR_RED("Sent ack after copy for sequence_num #" << sequence_num));
}

/**
 * @brief      Sends a processed data packet to data write.
 *
 * @param      pd    A processeddata protobuf object.
 */
void DSPCore::send_processed_data(processeddata::ProcessedData &pd)
{
  std::string p_msg_str;
  pd.SerializeToString(&p_msg_str);

  SEND_REPLY(*data_socket, sig_options.get_dw_dsp_identity(), p_msg_str);

  DEBUG_MSG(COLOR_RED("Send processed data to data_write for sequence #" << sequence_num));
}


/**
 * @brief      Starts the timing before the GPU kernels execute.
 *
 */
void DSPCore::start_decimate_timing()
{
  gpuErrchk(hipEventRecord(kernel_start, stream));
  gpuErrchk(hipEventRecord(mem_transfer_end,stream));
}

/**
 * @brief      Adds the callback to the CUDA stream to acknowledge the RF samples have been copied.
 *
 */
void DSPCore::initial_memcpy_callback()
{
  gpuErrchk(hipStreamAddCallback(stream, initial_memcpy_callback_handler, this, 0));
}


/**
 * @brief      Gets the device pointer to the RF samples.
 *
 * @return     The RF samples device pointer.
 */
hipComplex* DSPCore::get_rf_samples_p(){
  return rf_samples_d;
}

/**
 * @brief      Gets the host pointer to the RF samples.
 *
 * @return     The rf samples host pointer.
 */
std::vector<hipComplex> DSPCore::get_rf_samples_h() {
  return rf_samples_h;
}

/**
 * @brief      Gets the device pointer to the receive frequencies.
 *
 * @return     The frequencies device pointer.
 */
double* DSPCore::get_frequencies_p() {
  return freqs_d;
}

/**
 * @brief      Gets the device pointer to the first stage bandpass filters.
 *
 * @return     The first stage bandpass filters device pointer.
 */
hipComplex* DSPCore::get_first_stage_bp_filters_p(){
  return first_stage_bp_filters_d;
}

/**
 * @brief      Gets the device pointer to the second stage filters.
 *
 * @return     The second stage filter device pointer.
 */
hipComplex* DSPCore::get_second_stage_filter_p(){
  return second_stage_filter_d;
}

/**
 * @brief      Gets the device pointer to the third stage filters.
 *
 * @return     The third stage filter device pointer.
 */
hipComplex* DSPCore::get_third_stage_filter_p(){
  return third_stage_filter_d;
}

/**
 * @brief      Gets the device pointer to the fourth stage filters.
 *
 * @return     The fourth stage filter device pointer.
 */
hipComplex* DSPCore::get_fourth_stage_filter_p(){
  return fourth_stage_filter_d;
}

/**
 * @brief      Gets the device pointer to output of the first stage decimation.
 *
 * @return     The first stage output device pointer.
 */
hipComplex* DSPCore::get_first_stage_output_p(){
  return first_stage_output_d;
}

/**
 * @brief      Gets the device pointer to output of the second stage decimation.
 *
 * @return     The second stage output device pointer.
 */
hipComplex* DSPCore::get_second_stage_output_p(){
  return second_stage_output_d;
}

/**
 * @brief      Gets the device pointer to output of the third stage decimation.
 *
 * @return     The third stage output device pointer.
 */
hipComplex* DSPCore::get_third_stage_output_p(){
  return third_stage_output_d;
}

/**
 * @brief      Gets the device pointer to output of the fourth stage decimation.
 *
 * @return     The fourth stage output device pointer.
 */
hipComplex* DSPCore::get_fourth_stage_output_p(){
  return fourth_stage_output_d;
}

/**
 * @brief      Gets the host pointer to the output samples.
 *
 * @return     The host output pointer.
 */
hipComplex* DSPCore::get_host_output_h() {
  return host_output_h;
}

/**
 * @brief      Get the vector of host side frequencies.
 *
 * @return     The receive freqs vector.
 */
std::vector<double> DSPCore::get_rx_freqs()
{
  return rx_freqs;
}
/**
 * @brief      Gets the CUDA stream this DSPCore's work is associated to.
 *
 * @return     The CUDA stream.
 */
hipStream_t DSPCore::get_cuda_stream(){
  return stream;
}

/**
 * @brief      Gets the total GPU process timing in milliseconds.
 *
 * @return     The total process timing.
 */
float DSPCore::get_total_timing()
{
  return total_process_timing_ms;
}

/**
 * @brief      Gets the total decimation timing in milliseconds.
 *
 * @return     The decimation timing.
 */
float DSPCore::get_decimate_timing()
{
  return decimate_kernel_timing_ms;
}

/**
 * @brief      Gets the host pointer for first stage output.
 *
 * @return     The first stage output host pointer.
 */
hipComplex* DSPCore::get_first_stage_output_h()
{
  return first_stage_output_h;
}

/**
 * @brief      Gets the host pointer for the second stage output.
 *
 * @return     The second stage output host pointer.
 */
hipComplex* DSPCore::get_second_stage_output_h()
{
  return second_stage_output_h;
}

/**
 * @brief      Gets the host pointer for the third stage output.
 *
 * @return     The third stage output host pointer.
 */
hipComplex* DSPCore::get_third_stage_output_h()
{
  return third_stage_output_h;
}

/**
 * @brief      Gets the host pointer for the fourth stage output.
 *
 * @return     The fourth stage output host pointer.
 */
hipComplex* DSPCore::get_fourth_stage_output_h()
{
  return fourth_stage_output_h;
}

/**
 * @brief      Gets the number of antennas.
 *
 * @return     The number of antennas.
 */
uint32_t DSPCore::get_num_antennas()
{
  return num_antennas;
}

/**
 * @brief      Gets the number of rf samples.
 *
 * @return     The number of rf samples.
 */
uint32_t DSPCore::get_num_rf_samples()
{
  return num_rf_samples;
}

/**
 * @brief      Gets the number first stage samples per antenna.
 *
 * @return     The number first stage samples per antenna.
 */
uint32_t DSPCore::get_num_first_stage_samples_per_antenna()
{
  return num_first_stage_samples_per_antenna;
}

/**
 * @brief      Gets the number second stage samples per antenna.
 *
 * @return     The number second stage samples per antenna.
 */
uint32_t DSPCore::get_num_second_stage_samples_per_antenna()
{
  return num_second_stage_samples_per_antenna;
}

/**
 * @brief      Gets the number third stage samples per antenna.
 *
 * @return     The number third stage samples per antenna.
 */
uint32_t DSPCore::get_num_third_stage_samples_per_antenna()
{
  return num_third_stage_samples_per_antenna;
}

/**
 * @brief      Gets the number fourth stage samples per antenna.
 *
 * @return     The number fourth stage samples per antenna.
 */
uint32_t DSPCore::get_num_fourth_stage_samples_per_antenna()
{
  return num_fourth_stage_samples_per_antenna;
}

/**
 * @brief      Gets the sequence number.
 *
 * @return     The sequence number.
 */
uint32_t DSPCore::get_sequence_num()
{
  return sequence_num;
}

/**
 * @brief      Gets the rx sample rate.
 *
 * @return     The rx sampling rate (samples per second).
 */
double DSPCore::get_rx_rate()
{
  return rx_rate;
}

/**
 * @brief      Gets the output sample rate.
 *
 * @return     The output decimated and filtered rate (samples per second).
 */
double DSPCore::get_output_sample_rate()
{
  return output_sample_rate;
}

/**
 * @brief     Gets the vector of beam phases.
 *
 * @return    The beam phases.
 */
std::vector<hipComplex> DSPCore::get_beam_phases()
{
  return beam_phases;
}

/**
 * @brief     Gets the vector of beam direction counts for each RX frequency.
 *
 * @return    The beam direction counts.
 */
std::vector<uint32_t> DSPCore::get_beam_direction_counts()
{
  return beam_direction_counts;
}

/**
 * @brief     Gets the name of the shared memory section.
 *
 * @return    The shared memory name string.
 */
std::string DSPCore::get_shared_memory_name()
{
  return shm.get_region_name();
}

/**
 * @brief      Gets the driver initialization timestamp.
 *
 * @return     The driver initialization timestamp.
 */
double DSPCore::get_driver_initialization_time()
{
  return driver_initialization_time;
}

/**
 * @brief      Gets the sequence start timestamp.
 *
 * @return     The sequence start timestamp.
 */
double DSPCore::get_sequence_start_time()
{
  return sequence_start_time;
}

/**
 * @brief      Gets the vector of slice identifiers.
 *
 * @return     The vector of slice identifiers.
 */
std::vector<uint32_t> DSPCore::get_slice_ids()
{
  return slice_ids;
}
