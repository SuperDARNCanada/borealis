#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <zmq.hpp> // REVIEW #4 Need to explain what we use from this lib in our general documentation
#include <thread>
#include <complex>
#include <iostream>
#include <fstream>
#include <chrono>
#include <stdint.h>
#include <signal.h>
#include <cstdlib>
#include <math.h>
#include <numeric>
#include <functional>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include "utils/protobuf/rxsamplesmetadata.pb.h"
#include "utils/protobuf/sigprocpacket.pb.h"
#include "utils/driver_options/driveroptions.hpp"
#include "utils/signal_processing_options/signalprocessingoptions.hpp"
#include "utils/shared_memory/shared_memory.hpp"
#include "utils/shared_macros/shared_macros.hpp"
#include "utils/zmq_borealis_helpers/zmq_borealis_helpers.hpp"
#include "dsp.hpp"
#include "filtering.hpp"
#include "decimate.hpp"


int main(int argc, char **argv){
  GOOGLE_PROTOBUF_VERIFY_VERSION; // Verifies that header and lib are same version.

  //TODO(keith): verify config options.
  auto sig_options = SignalProcessingOptions();

  zmq::context_t context(1); // 1 is context num. Only need one per program as per examples
  auto identities = {sig_options.get_dsp_radctrl_identity(),
                   sig_options.get_dsp_driver_identity(),
                   sig_options.get_dsp_exphan_identity()};

  auto sockets_vector = create_sockets(context, identities, sig_options.get_router_address());

  zmq::socket_t &dsp_to_radar_control = sockets_vector[0];
  zmq::socket_t &dsp_to_driver = sockets_vector[1];
  zmq::socket_t &dsp_to_experiment_handler = sockets_vector[2];

  auto gpu_properties = get_gpu_properties();
  print_gpu_properties(gpu_properties);

  SharedMemoryHandler shrmem(sig_options.get_ringbuffer_name());
  std::vector<hipComplex*> ringbuffer_ptrs_start;

  std::vector<std::vector<float>> filter_taps;
  Filtering filters;

  std::vector<uint32_t> dm_rates;
  double rx_rate;
  uint32_t total_antennas;
  double output_sample_rate;

  auto first_time = true;
  for(;;) {

    //Receive first packet from radar control
    //auto message =  std::string("Need metadata");
    //SEND_REQUEST(dsp_to_radar_control, sig_options.get_radctrl_dsp_identity(), message);
    auto reply = RECV_REPLY(dsp_to_radar_control, sig_options.get_radctrl_dsp_identity());

    sigprocpacket::SigProcPacket sp_packet;
    if (sp_packet.ParseFromString(reply) == false){
      //TODO(keith): handle error
    }

    if (first_time) {
      total_antennas = sig_options.get_main_antenna_count() +
                  sig_options.get_interferometer_antenna_count();

      // First time - set up rx rate and filters.
      rx_rate = sp_packet.rxrate(); //Hz
      output_sample_rate = sp_packet.output_sample_rate(); //Hz

      for (uint32_t i=0; i<sp_packet.decimation_stages_size(); i++) {
        dm_rates.push_back(sp_packet.decimation_stages(i).dm_rate());

        std::vector<float> taps(sp_packet.decimation_stages(i).filter_taps().begin(),
        sp_packet.decimation_stages(i).filter_taps().end());
        filter_taps.push_back(taps);
      }

      RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Decimation rates: ");
      for (auto &rate : dm_rates) {
        RUNTIME_MSG("   " << rate);
      }

      filters = Filtering(filter_taps);

      RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Number of taps per stage: ");
      for (auto &taps : filter_taps) {
        RUNTIME_MSG("   " << COLOR_MAGENTA(taps.size()));
      }

      RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") <<
                  "Number of taps per stage after padding: ");
      for (auto &taps : filters.get_unmixed_filter_taps()) {
        RUNTIME_MSG("   " << COLOR_MAGENTA(taps.size()));
      }
    } // if (first_time)

    //Then receive first packet from driver
    auto message = std::string("Need data to process");
    SEND_REQUEST(dsp_to_driver, sig_options.get_driver_dsp_identity(), message);
    reply = RECV_REPLY(dsp_to_driver, sig_options.get_driver_dsp_identity());

    rxsamplesmetadata::RxSamplesMetadata rx_metadata;
    if (rx_metadata.ParseFromString(reply) == false) {
      //TODO(keith): handle error
    }

    if (first_time) {
      // First time - set up memory
      shrmem.open_shr_mem();
      if (rx_metadata.ringbuffer_size() == 0) {
        //TODO(keith): handle error
      }
      for(uint32_t i=0; i<total_antennas; i++){
        auto ptr = static_cast<hipComplex*>(shrmem.get_shrmem_addr()) +
                                              (i * rx_metadata.ringbuffer_size());
        ringbuffer_ptrs_start.push_back(ptr);
      }
      first_time = false;
    }

    RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") << "Got driver request for sequence #"
      << COLOR_RED(rx_metadata.sequence_num()));

    //Verify driver and radar control packets align
    if (sp_packet.sequence_num() != rx_metadata.sequence_num()) {
      //TODO(keith): handle error
      RUNTIME_MSG(COLOR_MAGENTA("SIGNAL PROCESSING: ") <<"SEQUENCE NUMBER mismatch radar_control: "
        << COLOR_RED(sp_packet.sequence_num()) << " usrp_driver: "
        << COLOR_RED(rx_metadata.sequence_num()));
    }

    if (rx_metadata.rx_rate() != rx_rate) {
      //TODO handle error
    }

    //Parse needed packet values now
    if (sp_packet.rxchannel_size() == 0) {
      //TODO(keith): handle error
    }


    // Parse out the beam phases and other relevant info from the radar control signal proc packet.
    std::vector<hipComplex> beam_phases;
    std::vector<rx_slice> slice_info;

    for (uint32_t channel=0; channel<sp_packet.rxchannel_size(); channel++) {
      // In this case each channel is the info for a new RX frequency
      auto rx_channel = sp_packet.rxchannel(channel);

      auto rx_freq = rx_channel.rxfreq();
      auto slice_id = rx_channel.slice_id();
      auto num_ranges = rx_channel.num_ranges();
      auto first_range = rx_channel.first_range();
      auto range_sep = rx_channel.range_sep();
      auto beam_count = rx_channel.beam_directions_size();
      auto tau_spacing = rx_channel.tau_spacing();
      auto new_rx_slice = rx_slice(rx_freq, slice_id, num_ranges, beam_count, first_range, range_sep,
                                    tau_spacing);

      auto num_lags = sp_packet.rxchannel(channel).lags_size();
      for (uint32_t lag_counter=0; lag_counter<num_lags; lag_counter++) {
        auto lag_num = sp_packet.rxchannel(channel).lags(lag_counter).lag_num();
        auto pulse_1 = sp_packet.rxchannel(channel).lags(lag_counter).pulse_1();
        auto pulse_2 = sp_packet.rxchannel(channel).lags(lag_counter).pulse_2();
        new_rx_slice.lags.push_back({pulse_1, pulse_2, lag_num});
      }
      slice_info.push_back(new_rx_slice);



      // We are going to use two intermediate vectors here to rearrange the phase data so that
      // all M data comes first, followed by all I data. This way can we directly treat each
      // block of memory as a matrix for beamforming the individual arrays.
      std::vector<hipComplex> main_phases;
      std::vector<hipComplex> intf_phases;

      for (uint32_t beam_num=0; beam_num<rx_channel.beam_directions_size(); beam_num++) {
        // Go through each beam now and add the phases for each antenna to a vector.
        auto beam = rx_channel.beam_directions(beam_num);

        for(uint32_t phase_num=0; phase_num<beam.phase_size(); phase_num++) {
          auto phase = beam.phase(phase_num);
          hipComplex new_angle;
          new_angle.x = phase.real_phase();
          new_angle.y = phase.imag_phase();

          if (phase_num < sig_options.get_main_antenna_count()) {
            main_phases.push_back(new_angle);
          }
          else {
            intf_phases.push_back(new_angle);
          }
        }
      }

      // Combine the separated antenna phases back into a flat vector.
      for (auto &phase : main_phases) {
        beam_phases.push_back(phase);
      }

      for (auto &phase : intf_phases) {
        beam_phases.push_back(phase);
      }
    }

    std::vector<double> rx_freqs;
    for (auto &s : slice_info) {
      rx_freqs.push_back(s.rx_freq);
    }
    TIMEIT_IF_TRUE_OR_DEBUG(false, "   NCO mix timing: ",
      [&]() {
        filters.mix_first_stage_to_bandpass(rx_freqs,rx_rate);
      }()
    );

    auto complex_taps = filters.get_mixed_filter_taps();

    DSPCore *dp = new DSPCore(std::ref(context), sig_options, sp_packet.sequence_num(),
                             rx_rate, output_sample_rate,
                             filter_taps, beam_phases,
                             rx_metadata.initialization_time(),
                             rx_metadata.sequence_start_time(),
                             rx_metadata.gps_locked(), rx_metadata.gps_to_system_time_diff(),
                             rx_metadata.agc_status_bank_h(), rx_metadata.lp_status_bank_h(),
                             rx_metadata.agc_status_bank_l(), rx_metadata.lp_status_bank_l(),
                             dm_rates, slice_info);

    if (rx_metadata.numberofreceivesamples() == 0){
      //TODO(keith): handle error for missing number of samples.
    }

    //We need to sample early to account for propagating samples through filters. The number of
    //required early samples is equal to adding half the filter length of each stage, starting with
    //the last stage so that the center point of the filter(point of highest gain) aligns with the
    //center of the pulse. This is the exact number of extra samples needed so that the output
    //data after decimation correctly aligns to the center of the first pulse.
    int64_t extra_samples = 0;

    for (int32_t i=dm_rates.size()-1; i>=0; i--) {
      extra_samples = (extra_samples * dm_rates[i]) + (filter_taps[i].size()/2);
    }


    auto total_dm_rate = std::accumulate(dm_rates.begin(), dm_rates.end(), 1,
                                            std::multiplies<int64_t>());

    auto samples_needed = rx_metadata.numberofreceivesamples() + 2 * extra_samples;
    samples_needed = uint32_t(std::ceil(float(samples_needed)/float(total_dm_rate)) *
                              total_dm_rate);
    auto total_samples = samples_needed * total_antennas;

    DEBUG_MSG("   Total samples in data message: " << total_samples);

    dp->allocate_and_copy_frequencies(rx_freqs.data(), rx_freqs.size());

    auto offset_to_first_rx_sample = sp_packet.offset_to_first_rx_sample();
    //offset_to_first_rx_sample = 0;
    dp->allocate_and_copy_rf_samples(total_antennas, samples_needed, extra_samples,
                                offset_to_first_rx_sample,
                                rx_metadata.initialization_time(),
                                rx_metadata.sequence_start_time(),
                                rx_metadata.ringbuffer_size(), ringbuffer_ptrs_start);

    dp->allocate_and_copy_bandpass_filters(complex_taps[0].data(), complex_taps[0].size());

    auto num_output_samples_per_antenna = samples_needed / dm_rates[0];
    auto total_output_samples_1 = rx_freqs.size() * num_output_samples_per_antenna *
                                   total_antennas;

    dp->allocate_output(total_output_samples_1);

    dp->initial_memcpy_callback();

    auto last_filter_output = dp->get_last_filter_output_d();
    call_decimate<DecimationType::bandpass>(dp->get_rf_samples_p(),
      last_filter_output, dp->get_bp_filters_p(), dm_rates[0],
      samples_needed, complex_taps[0].size(),
      rx_freqs.size(), total_antennas, rx_rate, dp->get_frequencies_p(),
      "Bandpass stage of decimation", dp->get_cuda_stream());


    std::vector<uint32_t> samples_per_antenna(complex_taps.size());
    std::vector<uint32_t> total_output_samples(complex_taps.size());

    samples_per_antenna[0] = num_output_samples_per_antenna;
    total_output_samples[0] = total_output_samples_1;

    // When decimating, we go from one set of samples for each antenna in the first stage
    // to multiple sets of reduced samples for each frequency in further stages. Output samples are
    // grouped by frequency with all samples for each antenna following each other
    // before samples of another frequency start. In the first stage need a filter for each
    // frequency, but in the next stages we only need one filter for all data sets.
    hipComplex* prev_output = last_filter_output;
    for (uint32_t i=1; i<complex_taps.size(); i++) {
      samples_per_antenna[i] = samples_per_antenna[i-1]/dm_rates[i];
      total_output_samples[i] = rx_freqs.size() * samples_per_antenna[i] * total_antennas;

      dp->allocate_and_copy_lowpass_filter(complex_taps[i].data(), complex_taps[i].size());
      dp->allocate_output(total_output_samples[i]);

      auto allocated_lp_filter = dp->get_last_lowpass_filter_d();
      last_filter_output = dp->get_last_filter_output_d();

      call_decimate<DecimationType::lowpass>(prev_output, last_filter_output, allocated_lp_filter,
        dm_rates[i], samples_per_antenna[i-1], complex_taps[i].size(), rx_freqs.size(),
        total_antennas, rx_rate, dp->get_frequencies_p(), " stage of decimation",
        dp->get_cuda_stream());

      prev_output = last_filter_output;
    }

    dp->cuda_postprocessing_callback(total_antennas, samples_needed, samples_per_antenna,
                                      total_output_samples);

  } //for(;;)
}
