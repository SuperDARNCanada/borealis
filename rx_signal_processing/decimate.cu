#include "hip/hip_runtime.h"
/*
Copyright 2017 SuperDARN Canada

See LICENSE for details

This file contains the CUDA code used to process the large amount of data
involved with radar receive side processing.
*/

#include <hip/hip_complex.h> //hipComplex type and all hipCmulf/hipCaddf functions.
#include <iostream>
#include <stdint.h>
#include "decimate.hpp"
//This keeps the contained functions local to this file.
namespace {
  /**
   * @brief      Creates a new set of grid dimensions for a bandpass decimate CUDA kernel.
   *
   * @param[in]  num_samples   Number of input samples.
   * @param[in]  dm_rate       Decimation rate.
   * @param[in]  num_antennas  Number of antennas for which there are samples.
   *
   * @return     New grid dimensions for the kernel.
   */
  dim3 create_bandpass_grid(uint32_t num_samples, uint32_t dm_rate, uint32_t num_antennas)
  {
    auto num_blocks_x = num_samples/dm_rate;
    auto num_blocks_y = num_antennas;
    auto num_blocks_z = 1;
    DEBUG_MSG(COLOR_BLUE("Decimate: ") << "    Grid size: " << num_blocks_x << " x "
      << num_blocks_y << " x "<< num_blocks_z);
    dim3 dimGrid(num_blocks_x,num_blocks_y,num_blocks_z);

    return dimGrid;
  }

  /**
   * @brief      Creates a new set of block dimensions for a bandpass decimate CUDA kernel.
   *
   * @param[in]  num_total_taps       Number of taps per stage.
   * @param[in]  num_freqs            Number of receive frequencies.
   *
   * @return     New block dimensions for the kernel.
   */
  dim3 create_bandpass_block(uint32_t num_total_taps, uint32_t num_freqs)
  {
    auto num_threads_x = num_total_taps/num_freqs;
    auto num_threads_y = num_freqs;
    auto num_threads_z = 1;
    DEBUG_MSG(COLOR_BLUE("Decimate: ") << "    Block size: " << num_threads_x << " x "
      << num_threads_y << " x " << num_threads_z);
    dim3 dimBlock(num_threads_x,num_threads_y,num_threads_z);

    return dimBlock;
  }

  /**
   * @brief      Creates a new set of grid dimensions for a lowpass decimate CUDA kernel.
   *
   * @param[in]  num_samples   Number of input samples in a frequency dataset.
   * @param[in]  dm_rate       Decimation rate.
   * @param[in]  num_antennas  Number of antennas for which there are samples.
   * @param[in]  num_freqs     Number of receive frequencies.
   *
   * @return     New grid dimensions for the kernel.
   */
  dim3 create_lowpass_grid(uint32_t num_samples, uint32_t dm_rate, uint32_t num_antennas,
                           uint32_t num_freqs)
  {
    auto num_blocks_x = num_samples/dm_rate;
    auto num_blocks_y = num_antennas;
    auto num_blocks_z = num_freqs;
    DEBUG_MSG(COLOR_BLUE("Decimate: ") << "    Grid size: " << num_blocks_x << " x "
      << num_blocks_y << " x "<< num_blocks_z);
    dim3 dimGrid(num_blocks_x,num_blocks_y,num_blocks_z);

    return dimGrid;
  }

  /**
   * @brief      Creates a new set of block dimensions for a lowpass decimate CUDA kernel.
   *
   * @param[in]  num_total_taps  Number of taps per stage.
   *
   * @return     New block dimensions for the kernel.
   */
  dim3 create_lowpass_block(uint32_t num_total_taps)
  {
    auto num_threads_x = num_total_taps;
    auto num_threads_y = 1;
    auto num_threads_z = 1;
    DEBUG_MSG(COLOR_BLUE("Decimate: ") << "    Block size: " << num_threads_x << " x "
      << num_threads_y << " x " << num_threads_z);
    dim3 dimBlock(num_threads_x,num_threads_y,num_threads_z);

    return dimBlock;
  }
}

/**
 * @brief      Overloads __shfl_down to handle hipComplex.
 *
 * @param[in]  var      hipComplex value to shuffle.
 * @param[in]  srcLane  Relative lane from within the warp that should shuffle its variable down.
 * @param[in]  width    Section of the warp to shuffle. Defaults to full warp size.
 *
 * @return     Shuffled hipComplex variable.
 *
 * __shfl can only shuffle 4 bytes at time. This overload utilizes a trick similar to the below
 * link in order to shuffle 8 byte values.
 * https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
 * http://docs.nvidia.com/cuda/cuda-c-programming-guide/#warp-shuffle-functions
 */
__device__ inline hipComplex __shfl_down_sync(hipComplex var, unsigned int srcLane, int width=32){
  float2 a = *reinterpret_cast<float2*>(&var);
  a.x = __shfl_down_sync(0xFFFFFFFF, a.x, srcLane, width);
  a.y = __shfl_down_sync(0xFFFFFFFF, a.y, srcLane, width);
  return *reinterpret_cast<hipComplex*>(&a);
}

/**
 * @brief      Performs a parallel reduction to sum a series of values together.
 *
 * @param      data        A pointer to a set of hipComplex data to reduce.
 * @param[in]  tap_offset  The offset into the data from which to pull values.
 *
 * @return     Final sum after reduction.
 *
 * NVIDIA supplies many versions of optimized parallel reduction. This is a slightly modified
 * version of reduction #5 from NVIDIA examples.
 * /usr/local/cuda/samples/6_Advanced/reduction
 */
__device__ hipComplex parallel_reduce(hipComplex* data, uint32_t tap_offset) {
  auto filter_tap_num = threadIdx.x;
  auto num_filter_taps = blockDim.x;
  hipComplex total_sum = data[tap_offset];

  if ((num_filter_taps >= 1024) && (filter_tap_num < 512))
  {
    total_sum = hipCaddf(total_sum,data[tap_offset  + 512]);
    data[tap_offset] = total_sum;
  }

  __syncthreads();

  if ((num_filter_taps >= 512) && (filter_tap_num < 256))
  {
    total_sum = hipCaddf(total_sum,data[tap_offset  + 256]);
    data[tap_offset] = total_sum;
  }

  __syncthreads();

  if ((num_filter_taps >= 256) && (filter_tap_num < 128))
  {
    total_sum = hipCaddf(total_sum, data[tap_offset + 128]);
    data[tap_offset] = total_sum;
  }

   __syncthreads();

  if ((num_filter_taps >= 128) && (filter_tap_num <  64))
  {
    total_sum = hipCaddf(total_sum, data[tap_offset  +  64]);
    data[tap_offset] = total_sum;
  }

  __syncthreads();

  if ( filter_tap_num < 32 )
  {
    // Fetch final intermediate sum from 2nd warp
    if (num_filter_taps >=  64) total_sum = hipCaddf(total_sum, data[tap_offset + 32]);
    // Reduce final warp using shuffle
    // http://docs.nvidia.com/cuda/cuda-c-programming-guide/#built-in-variables
    // __shfl_down is used an optimization in the final warp to simulatenously move
    // values from upper threads to lower threads without needing __syncthreads().
    for (int offset = warpSize/2; offset > 0; offset /= 2)
    {
      total_sum = hipCaddf(total_sum,__shfl_down_sync(total_sum, offset));
    }
  }

  return total_sum;
}

/**
 * @brief      hipComplex version of exponential function.
 *
 * @param[in]  z     Complex number.
 *
 * @return     Complex exponential of input.
 */
__device__ __forceinline__ hipComplex _exp (hipComplex z)
{
    hipComplex res;
    float t = expf(z.x);
    sincosf(z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

/**
 * @brief      Performs decimation using bandpass filters on a set of input RF samples if the total
 *             number of filter taps for all filters is less than 1024.
 *
 * @param[in]  original_samples     A pointer to original input samples from each antenna to
 *                                  decimate.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each
 *                                  frequency after decimation.
 * @param[in]  filter_taps          A pointer to one or more filters needed for each frequency.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 * @param[in]  F_s                  The sampling frequency in hertz.
 * @param[in]  freqs                A pointer to the frequencies used in mixing.
 *
 * This function performs a parallel version of filtering+downsampling on the GPU to be able
 * process data in realtime. This algorithm will use 1 GPU thread per filter tap if there are less
 * than 1024 taps for all filters combined. Only works with power of two length filters, or a
 * filter that is zero padded to a power of two in length. This algorithm takes
 * a single set of wide band samples from the USRP driver, and produces an output data set for each
 * RX frequency. The phase of each output sample is corrected to after decimating via modified
 * Frerking method.
 *
 *   gridDim.x - Total number of output samples there will be after decimation.
 *   gridDim.y - Total number of antennas.
 *
 *   blockIdx.x - Decimated output sample index.
 *   blockIdx.y - Antenna index.
 *
 *   blockDim.x - Number of filter taps in the lowpass filter.
 *   blockDim.y - Total number of filters. Corresponds to total receive frequencies.
 *
 *   threadIdx.x - Filter tap index.
 *   threadIdx.y - Filter index.
 */
__global__ void bandpass_decimate1024(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna, double F_s, double *freqs) {

  // Since number of filter taps is calculated at runtime and we do not want to hardcode
  // values, the shared memory can be dynamically initialized at invocation of the kernel.
  // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared

  extern __shared__ hipComplex filter_products[];

  auto antenna_num = blockIdx.y;
  auto antenna_offset = antenna_num * samples_per_antenna;

  auto dec_sample_num = blockIdx.x;
  auto dec_sample_offset = dec_sample_num * dm_rate;

  auto tap_offset = threadIdx.y * blockDim.x + threadIdx.x;

  // If an offset should extend past the length of samples per antenna
  // then zeroes are used as to not segfault or run into the next buffer.
  // output samples convolved with these zeroes will be discarded after
  // the complete process as to not introduce edge effects.
  hipComplex sample;
  if ((dec_sample_offset + threadIdx.x) >= samples_per_antenna) {
    sample = make_hipComplex(0.0f,0.0f);
  }
  else {
    auto final_offset = antenna_offset + dec_sample_offset + threadIdx.x;
    sample = original_samples[final_offset];
  }

  filter_products[tap_offset] = hipCmulf(sample,filter_taps[tap_offset]);
  // Synchronizes all threads in a block, meaning 1 output sample per rx freq
  // is ready to be calculated with the parallel reduce
  __syncthreads();

  auto calculated_output_sample = parallel_reduce(filter_products, tap_offset);

  // When decimating, we go from one set of samples for each antenna
  // to multiple sets of reduced samples for each frequency. Output samples are
  // grouped by frequency with all samples for each antenna following each other
  // before samples of another frequency start.
  if (threadIdx.x == 0) {

    //Correct phase after filtering using modified Frerking technique.
    auto freq_idx = threadIdx.y;
    auto unwrapped_phase = 2.0 * M_PI * (freqs[freq_idx]/F_s) * dec_sample_num * dm_rate;
    auto phase = fmod(unwrapped_phase, 2.0 * M_PI);
    auto filter_phase = _exp(make_hipComplex(0.0f, 1 * phase));
    calculated_output_sample = hipCmulf(calculated_output_sample,filter_phase);

    antenna_offset = antenna_num * gridDim.x;
    auto total_antennas = gridDim.y;
    auto freq_offset = threadIdx.y * gridDim.x * total_antennas;
    auto total_offset = freq_offset + antenna_offset + dec_sample_num;
    decimated_samples[total_offset] = calculated_output_sample;

  }
}

/**
 * @brief      Performs decimation using bandpass filters on a set of input RF samples if the total
 *             number of filter taps for all filters is less than 2048.
 *
 * @param[in]  original_samples     A pointer to original input samples from each antenna to
 *                                  decimate.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  after decimation.
 * @param[in]  filter_taps          A pointer to one or more filters needed for each frequency.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 * @param[in]  F_s                  The sampling frequency in hertz.
 * @param[in]  freqs                A pointer to the frequencies used in mixing.
 *
 * This function performs a parallel version of filtering+downsampling on the GPU to be able process
 * data in realtime. This algorithm will use 1 GPU thread to process two filter taps if there are
 * less than 2048 taps for all filters combined. Intended to be used if there are more than 1024
 * total threads, as that is the max block size possible for CUDA. Only works with power of two
 * length filters, or a filter that is zero padded to a power of two in length. This algorithm takes
 * a single set of wide band samples from the USRP driver, and produces a output data set for each
 * RX frequency.
 *
 *   gridDim.x - Total number of output samples there will be after decimation.
 *   gridDim.y - Total number of antennas.
 *
 *   blockIdx.x - Decimated output sample index.
 *   blockIdx.y - Antenna index.
 *
 *   blockDim.x - Number of filter taps in each filter / 2.
 *   blockDim.y - Total number of filters. Corresponds to total receive frequencies.
 *
 *   threadIdx.x - Every second filter tap index.
 *   threadIdx.y - Filter index.
 */
__global__ void bandpass_decimate2048(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna, double F_s, double *freqs)
{

  // Since number of filter taps is calculated at runtime and we do not want to hardcode
  // values, the shared memory can be dynamically initialized at invocation of the kernel.
  // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared

  extern __shared__ hipComplex filter_products[];

  auto antenna_num = blockIdx.y;
  auto antenna_offset = antenna_num * samples_per_antenna;

  auto dec_sample_num = blockIdx.x;
  auto dec_sample_offset = dec_sample_num * dm_rate;

  auto tap_offset = threadIdx.y * blockDim.x + 2 * threadIdx.x;

  hipComplex sample_1;
  hipComplex sample_2;

  // If an offset should extend past the length of samples per antenna
  // then zeroes are used as to not segfault or run into the next buffer.
  // output samples convolved with these zeroes will be discarded after
  // the complete process as to not introduce edge effects.
  if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_antenna) {
    // the case both samples are out of bounds
    sample_1 = make_hipComplex(0.0,0.0);
    sample_2 = make_hipComplex(0.0,0.0);
  }
  else if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_antenna - 1) {
    // the case only one sample would be out of bounds
    auto final_offset = antenna_offset + dec_sample_offset + 2*threadIdx.x;
    sample_1 = original_samples[final_offset];
    sample_2 = make_hipComplex(0.0,0.0);
  }
  else {
    auto final_offset = antenna_offset + dec_sample_offset + 2*threadIdx.x;
    sample_1 = original_samples[final_offset];
    sample_2 = original_samples[final_offset+1];
  }

  filter_products[tap_offset] = hipCmulf(sample_1,filter_taps[tap_offset]);
  filter_products[tap_offset+1] = hipCmulf(sample_2, filter_taps[tap_offset+1]);

  // An additional add must happen first in this case since the parallel reduce will only
  // run on even data indices.
  filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],filter_products[tap_offset+1]);
  __syncthreads();

  auto calculated_output_sample = parallel_reduce(filter_products, tap_offset);

  // When decimating, we go from one set of samples for each antenna
  // to multiple sets of reduced samples for each frequency. Output samples are
  // grouped by frequency with all samples for each antenna following each other
  // before samples of another frequency start.
  if (threadIdx.x == 0) {

    //Correct phase after filtering using modified Frerking technique.
    auto freq_idx = threadIdx.y;
    auto unwrapped_phase = 2.0 * M_PI * (freqs[freq_idx]/F_s) * dec_sample_num * dm_rate;
    auto phase = fmod(unwrapped_phase, 2.0 * M_PI);
    auto filter_phase = _exp(make_hipComplex(0.0f, 1 * phase));
    calculated_output_sample = hipCmulf(calculated_output_sample,filter_phase);

    antenna_offset = antenna_num * gridDim.x;
    auto total_antennas = gridDim.y;
    auto freq_offset = threadIdx.y * gridDim.x * total_antennas;
    auto total_offset = freq_offset + antenna_offset + dec_sample_num;
    decimated_samples[total_offset] = calculated_output_sample;
  }
}


/**
 * @brief      This function wraps the bandpass_decimate1024 kernel so that it can be called from
 *             another file.
 *
 * @param[in]  original_samples     A pointer to original input samples from each antenna to
 *                                  decimate.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  after decimation.
 * @param[in]  filter_taps          A pointer to one or more filters needed for each frequency.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 * @param[in]  num_taps_per_filter  Number of taps per filter.
 * @param[in]  num_freqs            Number of receive frequencies.
 * @param[in]  num_antennas         Number of antennas for which there are samples.
 * @param[in]  F_s                  The original sampling frequency.
 * @param      freqs                A pointer to the frequencies being filtered.
 * @param[in]  stream               CUDA stream with which to associate the invocation of the
 *                                  kernel.
 */
void bandpass_decimate1024_wrapper(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna, uint32_t num_taps_per_filter, uint32_t num_freqs,
  uint32_t num_antennas, double F_s, double *freqs, hipStream_t stream) {

  //Allocate shared memory on device for all filter taps.
  auto shr_mem_taps = num_freqs * num_taps_per_filter * sizeof(hipComplex);
  DEBUG_MSG(COLOR_BLUE("Decimate: ") << "    Number of shared memory bytes: "<< shr_mem_taps);

  auto dimGrid = create_bandpass_grid(samples_per_antenna, dm_rate, num_antennas);
  auto dimBlock = create_bandpass_block(num_taps_per_filter,num_freqs);
  bandpass_decimate1024<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
        filter_taps, dm_rate, samples_per_antenna, F_s, freqs);

}




/**
 * @brief      This function wraps the bandpass_decimate2048 kernel so that it can be called from
 *             another file.
 *
 * @param[in]  original_samples     A pointer to original input samples from each antenna to
 *                                  decimate.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  after decimation.
 * @param[in]  filter_taps          A pointer to one or more filters needed for each frequency.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 * @param[in]  num_taps_per_filter  Number of taps per filter.
 * @param[in]  num_freqs            Number of receive frequencies.
 * @param[in]  num_antennas         Number of antennas for which there are samples.
 * @param[in]  F_s                  The original sampling frequency.
 * @param      freqs                A pointer to the frequencies being filtered.
 * @param[in]  stream               CUDA stream with which to associate the invocation of the
 *                                  kernel.
 */
void bandpass_decimate2048_wrapper(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna, uint32_t num_taps_per_filter, uint32_t num_freqs,
  uint32_t num_antennas, double F_s, double *freqs, hipStream_t stream) {

  //Allocate shared memory on device for all filter taps.
  auto shr_mem_taps = num_freqs * num_taps_per_filter * sizeof(hipComplex);
  DEBUG_MSG(COLOR_BLUE("Decimate: ") << "    Number of shared memory bytes: "<< shr_mem_taps);

  auto dimGrid = create_bandpass_grid(samples_per_antenna, dm_rate, num_antennas);
  auto dimBlock = create_bandpass_block(num_taps_per_filter/2, num_freqs);
  bandpass_decimate2048<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
    filter_taps, dm_rate, samples_per_antenna, F_s, freqs);
}

/**
 * @brief      Performs decimation using a lowpass filter on one or more sets of baseband samples
 * corresponding to each RX frequency. This algorithm works on filters with less that 1024 taps.
 *
 * @param[in]  original_samples     A pointer to input samples for one or more baseband datasets.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  dataset after decimation.
 * @param[in]  filter_taps          A pointer to a lowpass filter used for further decimation.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 *
 * This function performs a parallel version of filtering+downsampling on the GPU to be able
 * process data in realtime. This algorithm will use 1 GPU thread per filter tap if there are less
 * than 1024 taps for all filters combined. Only works with power of two length filters, or a
 * filter that is zero padded to a power of two in length. This algorithm takes one or more
 * baseband datasets corresponding to each RX frequency and filters each one using a single lowpass
 * filter before downsampling.
 *
 *   gridDim.x - The number of decimated output samples for one antenna in one frequency data set.
 *   gridDim.y - Total number of antennas.
 *   gridDim.z - Total number of frequency data sets.
 *
 *   blockIdx.x - Decimated output sample index.
 *   blockIdx.y - Antenna index.
 *   blockIdx.z - Frequency dataset index.
 *
 *   blockDim.x - Number of filter taps in the lowpass filter.

 *   threadIdx.x - Filter tap indices.
 */
__global__ void lowpass_decimate1024(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna) {

  // Since number of filter taps is calculated at runtime and we do not want to hardcode
  // values, the shared memory can be dynamically initialized at invocation of the kernel.
  // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared

  extern __shared__ hipComplex filter_products[];

  auto total_antennas = gridDim.y;

  auto data_set_idx = blockIdx.z;

  auto frequency_dataset_offset = data_set_idx * samples_per_antenna * total_antennas;

  auto antenna_num = blockIdx.y;
  auto antenna_offset = antenna_num * samples_per_antenna;

  auto dec_sample_num = blockIdx.x;
  auto dec_sample_offset = dec_sample_num * dm_rate;

  auto tap_offset = threadIdx.x;

  // If an offset should extend past the length of samples per antenna
  // then zeroes are used as to not segfault or run into the next buffer.
  // output samples convolved with these zeroes will be discarded after
  // the complete process as to not introduce edge effects.
  hipComplex sample;
  if ((dec_sample_offset + tap_offset) >= samples_per_antenna) {
    sample = make_hipComplex(0.0f,0.0f);
  }
  else {
    auto final_offset = frequency_dataset_offset + antenna_offset + dec_sample_offset + tap_offset;
    sample = original_samples[final_offset];
  }


  filter_products[tap_offset] = hipCmulf(sample,filter_taps[tap_offset]);
  // Synchronizes all threads in a block, meaning 1 output sample per rx freq
  // is ready to be calculated with the parallel reduce
  __syncthreads();

  auto calculated_output_sample = parallel_reduce(filter_products, tap_offset);

  // When decimating, we go from one set of samples for each antenna
  // to multiple sets of reduced samples for each frequency. Output samples are
  // grouped by frequency with all samples for each antenna following each other
  // before samples of another frequency start.
  if (threadIdx.x == 0) {
    auto num_output_samples_per_antenna = gridDim.x;
    frequency_dataset_offset = data_set_idx * num_output_samples_per_antenna * total_antennas;
    antenna_offset = antenna_num * num_output_samples_per_antenna;
    auto total_offset = frequency_dataset_offset + antenna_offset + dec_sample_num;
    decimated_samples[total_offset] = calculated_output_sample;
  }
}

/**
 * @brief      Performs decimation using a lowpass filter on one or more sets of baseband samples
 * corresponding to each RX frequency. This algorithm works on filters with less that 2048 taps.
 *
 * @param[in]  original_samples     A pointer to input samples for one or more baseband datasets.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  dataset after decimation.
 * @param[in]  filter_taps          A pointer to a lowpass filter used for further decimation.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in the original set of
 *                                  samples.
 *
 * This function performs a parallel version of filtering+downsampling on the GPU to be able process
 * data in realtime. This algorithm will use 1 GPU thread to process two filter taps if there are
 * less than 2048 taps for all filters combined. Intended to be used if there are more than 1024
 * total threads, as that is the max block size possible for CUDA. Only works with power of two
 * length filters, or a filter that is zero padded to a power of two in length. This algorithm takes
 * one or more baseband datasets corresponding to each RX frequency and filters each one using a
 * single lowpass filter before downsampling.
 *
 *   gridDim.x - The number of decimated output samples for one antenna in one frequency data set.
 *   gridDim.y - Total number of antennas.
 *   gridDim.z - Total number of frequency data sets.
 *
 *   blockIdx.x - Decimated output sample index.
 *   blockIdx.y - Antenna index.
 *   blockIdx.z - Frequency dataset index.
 *
 *   blockDim.x - Number of filter taps in the lowpass filter / 2.

 *   threadIdx.x - Every second filter tap index.
 */
__global__ void lowpass_decimate2048(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna)
{

  // Since number of filter taps is calculated at runtime and we do not want to hardcode
  // values, the shared memory can be dynamically initialized at invocation of the kernel.
  // http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared

  extern __shared__ hipComplex filter_products[];

  auto total_antennas = gridDim.y;

  auto data_set_idx = blockIdx.z;

  auto frequency_dataset_offset = data_set_idx * samples_per_antenna * total_antennas;

  auto antenna_num = blockIdx.y;
  auto antenna_offset = antenna_num * samples_per_antenna;

  auto dec_sample_num = blockIdx.x;
  auto dec_sample_offset = dec_sample_num * dm_rate;

  auto tap_offset = 2 * threadIdx.x;

  hipComplex sample_1;
  hipComplex sample_2;

  // If an offset should extend past the length of samples per antenna
  // then zeroes are used as to not segfault or run into the next buffer.
  // output samples convolved with these zeroes will be discarded after
  // the complete process as to not introduce edge effects.
  if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_antenna) {
    // the case both samples are out of bounds
    sample_1 = make_hipComplex(0.0,0.0);
    sample_2 = make_hipComplex(0.0,0.0);
  }
  else if ((dec_sample_offset + tap_offset) >= samples_per_antenna - 1) {
    // the case only one sample would be out of bounds
    auto final_offset = antenna_offset + dec_sample_offset + tap_offset;
    sample_1 = original_samples[final_offset];
    sample_2 = make_hipComplex(0.0,0.0);
  }
  else {
    auto final_offset = frequency_dataset_offset + antenna_offset + dec_sample_offset + tap_offset;
    sample_1 = original_samples[final_offset];
    sample_2 = original_samples[final_offset+1];
  }


  filter_products[tap_offset] = hipCmulf(sample_1,filter_taps[tap_offset]);
  filter_products[tap_offset+1] = hipCmulf(sample_2, filter_taps[tap_offset+1]);

  // An additional add must happen first in this case since the parallel reduce will only
  // run on even data indices.
  filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],filter_products[tap_offset+1]);
  __syncthreads();

  auto calculated_output_sample = parallel_reduce(filter_products, tap_offset);

  // When decimating, we go from one set of samples for each antenna
  // to multiple sets of reduced samples for each frequency. Output samples are
  // grouped by frequency with all samples for each antenna following each other
  // before samples of another frequency start.
  if (threadIdx.x == 0) {
    auto num_output_samples_per_antenna = gridDim.x;
    frequency_dataset_offset = data_set_idx * num_output_samples_per_antenna * total_antennas;
    antenna_offset = antenna_num * num_output_samples_per_antenna;
    auto total_offset = frequency_dataset_offset + antenna_offset + dec_sample_num;
    decimated_samples[total_offset] = calculated_output_sample;
  }
}

/**
 * @brief      This function wraps the lowpass_decimate1024 kernel so that it can be called from
 *             another file.
 *
 * @param[in]  original_samples     A pointer to one or more baseband frequency datasets.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  after decimation.
 * @param[in]  filter_taps          A pointer to one lowpass filter.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in each data set.
 * @param[in]  num_taps_per_filter  Number of taps per filter.
 * @param[in]  num_freqs            Number of receive frequency datasets.
 * @param[in]  num_antennas         Number of antennas for which there are samples.
 * @param[in]  stream               CUDA stream with which to associate the invocation of the
 *                                  kernel.
 */
void lowpass_decimate1024_wrapper(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna, uint32_t num_taps_per_filter, uint32_t num_freqs,
  uint32_t num_antennas, hipStream_t stream) {

  //Allocate shared memory on device for all filter taps.
  auto shr_mem_taps = num_freqs * num_taps_per_filter * sizeof(hipComplex);
  DEBUG_MSG(COLOR_BLUE("Decimate: ") << "    Number of shared memory bytes: "<< shr_mem_taps);

  auto dimGrid = create_lowpass_grid(samples_per_antenna, dm_rate, num_antennas, num_freqs);
  auto dimBlock = create_lowpass_block(num_taps_per_filter);
  lowpass_decimate1024<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples,
    decimated_samples, filter_taps, dm_rate, samples_per_antenna);
}

/**
 * @brief      This function wraps the lowpass_decimate2048 kernel so that it can be called from
 *             another file.
 *
 * @param[in]  original_samples     A pointer to one or more baseband frequency datasets.
 * @param[in]  decimated_samples    A pointer to a buffer to place output samples for each frequency
 *                                  after decimation.
 * @param[in]  filter_taps          A pointer to one lowpass filter.
 * @param[in]  dm_rate              Decimation rate.
 * @param[in]  samples_per_antenna  The number of samples per antenna in each data set.
 * @param[in]  num_taps_per_filter  Number of taps per filter.
 * @param[in]  num_freqs            Number of receive frequency datasets.
 * @param[in]  num_antennas         Number of antennas for which there are samples.
 * @param[in]  stream               CUDA stream with which to associate the invocation of the
 *                                  kernel.
 */
void lowpass_decimate2048_wrapper(hipComplex* original_samples,
  hipComplex* decimated_samples,
  hipComplex* filter_taps, uint32_t dm_rate,
  uint32_t samples_per_antenna, uint32_t num_taps_per_filter, uint32_t num_freqs,
  uint32_t num_antennas, hipStream_t stream) {

  //Allocate shared memory on device for all filter taps.
  auto shr_mem_taps = num_freqs * num_taps_per_filter * sizeof(hipComplex);
  DEBUG_MSG(COLOR_BLUE("Decimate: ") << "    Number of shared memory bytes: "<< shr_mem_taps);

  auto dimGrid = create_lowpass_grid(samples_per_antenna, dm_rate, num_antennas, num_freqs);
  auto dimBlock = create_lowpass_block(num_taps_per_filter/2);
  lowpass_decimate2048<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples,
    decimated_samples, filter_taps, dm_rate, samples_per_antenna);
}